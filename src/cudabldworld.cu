#include "hip/hip_runtime.h"
#ifndef _CUDA_BLD_WORLD_CU_
#define _CUDA_BLD_WORLD_CU_

#include <stdio.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>

#include "pnpstructs.h"
#include "pnpdebug.h"


#define FMUL __dmul_rn
#define FADD __dadd_rn
#define FMAF __dmaf_rn

#define BIGDISTANSE 10000

int checkCUDAError(const char* msg);

extern "C" int DoBldDielMapsOnCuda()
{
	return 1;
}

//BuildWorldNI *Builder,ContWorld* world,
extern "C" int BuildAtomsDielPreMapsOnCUDA0(GOAtomsStruct* atms,float *Displ)
{
	int iValue=atms->iDiel;
	int iBulkValue=atms->iDielBulk;
	float Rsmoth=atms->Rsmoth;
	int NAtoms=atms->Natoms;
	float *r[3]={atms->r[0],atms->r[1],atms->r[2]};
	float *R=atms->R;
	float *Surf=atms->Surf;
	int *Field=atms->iVtmp;
	DbgPrint2("GOAtoms::BuildPreMaps(iValue=%d iBulkValue=%d)\n"
			,iValue,iBulkValue);
// 	DbgPrint2("\t Rion=%f[grids] Rsmoth=%f[grids] \nDispl=[%f,%f,%f][grids,grids,grids]\n", Rion, Rsmoth, Displ[0], Displ[1], Displ[2]);
// 	DbgPrint2("\t r[0]=[%f,%f,%f] R=%f q=%f Natom=%d\n"
// 			,r[0][0],r[1][0],r[2][0],R[0],q[0],NAtoms);
// 	DbgPrint2("\t r[0]=[%f,%f,%f] R=%f q=%f Natom=%d\n"
// 			,r[0][NAtoms-1],r[1][NAtoms-1],r[2][NAtoms-1],R[NAtoms-1],q[NAtoms-1],NAtoms);
	int i,j,k,gridpoint,rint[3];
	float RSQ,RsmSQ,RtmpSQ,Rsm,Rtmp;
	float RleftSQ,RrightSQ;
	float rf[3];
	int iR,iRsm;
	int ix,iy,iz;
	float ftmp;
	float *vtmp;
	
	int GridSize[3]={atms->GS_X,atms->GS_Y,atms->GS_Z};
	int GSX=GridSize[0];
	int GSXY=GridSize[0]*GridSize[1];
	
	int start[3];
	int end[3];
	
	if(Rsmoth==0.0f)
	{
		DbgPrint0("Rsmoth==0.0f");
		for(i=0;i<NAtoms;i++)
		{
			iR=(int)(R[i]+0.5);
			RSQ=R[i]*R[i];
			for(k=0;k<3;k++){
				rf[k]=r[k][i]+Displ[k];
				rint[k]=(int)(rf[k]+0.5);
				start[k]=rint[k]-iR;
				end[k]=rint[k]+iR;
				if(start[k]<0)start[k]=0;
				if(end[k]>GridSize[k]-1)end[k]=GridSize[k]-1;
			}
			
			for(ix=start[0];ix<=end[0];ix++)
				for(iy=start[1];iy<=end[1];iy++)
					for(iz=start[2];iz<=end[2];iz++)
			{
				gridpoint=ix+iy*GSX+iz*GSXY;
				RtmpSQ=(rf[0]-ix)*(rf[0]-ix)+(rf[1]-iy)*(rf[1]-iy)+(rf[2]-iz)*(rf[2]-iz);
				vtmp=Surf+4*gridpoint;
				if(RtmpSQ<=RSQ)//RtmpSQ<=RSQ
				{
					Field[gridpoint]=iValue;
					vtmp[0]=-BIGDISTANSE;
				}
			}
		}
	}
	else
	{
		for(i=0;i<NAtoms;i++)
		{
			Rsm=R[i]+Rsmoth;
			iR=(int)(R[i]+0.5);
			iRsm=(int)(Rsm+1.0);
			RleftSQ=Rsm-0.707106781f;
			RrightSQ=Rsm+0.707106781f;
			RleftSQ*=RleftSQ;
			RrightSQ*=RrightSQ;
			RSQ=R[i]*R[i];
			RsmSQ=Rsm*Rsm;
			
			for(k=0;k<3;k++)
			{
				rf[k]=r[k][i]+Displ[k];
				rint[k]=(int)(rf[k]+0.5);
				start[k]=rint[k]-iRsm;
				end[k]=rint[k]+iRsm;
				if(start[k]<0)start[k]=0;
				if(end[k]>GridSize[k]-1)start[k]=GridSize[k]-1;
			}
			for(ix=start[0];ix<=end[0];ix++)
				for(iy=start[1];iy<=end[1];iy++)
					for(iz=start[2];iz<=end[2];iz++)
			{
				gridpoint=ix+iy*GSX+iz*GSXY;
				RtmpSQ=(rf[0]-ix)*(rf[0]-ix)+(rf[1]-iy)*(rf[1]-iy)+(rf[2]-iz)*(rf[2]-iz);
				vtmp=Surf+4*gridpoint;
				if(RtmpSQ<=RSQ)//RtmpSQ<=RSQ
				{
					Field[gridpoint]=iValue;
					vtmp[0]=-BIGDISTANSE;
					vtmp[3]=0.0;
				}
				else//RSQ<RtmpSQ
				{
					Rtmp=sqrt(RtmpSQ);
					if(RtmpSQ<=RsmSQ)//RSQ<RtmpSQ<RleftSQ
					{
						if(Field[gridpoint]==iBulkValue)
						{
							vtmp[0]=-BIGDISTANSE;
							Field[gridpoint]=-iValue;
							vtmp[3]=Rtmp-R[i];
						}
						else if(Field[gridpoint]<0&&Rtmp-R[i]<vtmp[3])
						{
							vtmp[0]=-BIGDISTANSE;
							Field[gridpoint]=-iValue;
							vtmp[3]=Rtmp-R[i];
						}
					}
					if(RleftSQ<=RtmpSQ&&RtmpSQ<=RrightSQ)//RleftSQ<=RtmpSQ<=RrightSQ
					{
						if(vtmp[0]>-100.0f)//e.i. is intersection
						{
							if(Rtmp-R[i]<vtmp[3])
							{
								ftmp=Rsm/Rtmp;
								vtmp[0]=rf[0]+ftmp*(ix-rf[0]);
								vtmp[1]=rf[1]+ftmp*(iy-rf[1]);
								vtmp[2]=rf[2]+ftmp*(iz-rf[2]);
								vtmp[3]=Rtmp-R[i];
							}
						}
						else if(Field[gridpoint]==iBulkValue)
						{
							ftmp=Rsm/Rtmp;
							vtmp[0]=rf[0]+ftmp*(ix-rf[0]);
							vtmp[1]=rf[1]+ftmp*(iy-rf[1]);
							vtmp[2]=rf[2]+ftmp*(iz-rf[2]);
							vtmp[3]=Rtmp-R[i];
						}
					}
				}
			}
		}
		/*if(MakePreRoll)
		{
			
			DbgPrint0("MakePreRoll\n");
			int lim=(int)(Rsmoth+0.5);
			int itmp=GSXYZ*4;
			int iSolAcFlag=1000000000;
			RsmSQ=Rsmoth*Rsmoth;
			for(i=0;i<itmp;i=i+4)
			{
				if(Surf[i]>-100.0)
				{
					vtmp=Surf+i;
					for(k=0;k<3;k++)
					{
						rint[k]=(int)(vtmp[k]+0.5);
						start[k]=rint[k]-lim;
						end[k]=rint[k]+lim;
						if(start[k]<0)start[k]=0;
						if(end[k]>GridSize[k]-1)end[k]=GridSize[k]-1;
					}
					for(ix=start[0];ix<=end[0];ix++)
						for(iy=start[1];iy<=end[1];iy++)
							for(iz=start[2];iz<=end[2];iz++)
					{
						gridpoint=ix+iy*GSX+iz*GSXY;
						RtmpSQ=(vtmp[0]-ix)*(vtmp[0]-ix)+(vtmp[1]-iy)*(vtmp[1]-iy)+(vtmp[2]-iz)*(vtmp[2]-iz);
						if(RtmpSQ<RsmSQ&&Field[gridpoint]<0)
						{
							Field[gridpoint]=iSolAcFlag;
						}
					}
				}
			}
			for(i=0;i<GSXYZ;i++)
			{
				if(Field[i]<0)
				{
					Field[i]=-Field[i];
				}
				else if(Field[i]==iSolAcFlag)
				{
					Field[i]=-iValue;
				}
			}
		}*/
	}
	return EXIT_SUCCESS;
}
__global__ void KBuildAtomsDielPreMaps(float *x,float *y,float *z,float *R, float *Surf,int *Field,int GS_X,int GS_Y,int GS_Z,float Rsmoth,float DisplX,float DisplY,float DisplZ,int iValue, int iBulkValue,int Natoms)
{
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	
	
	
	if(i<Natoms)
	{
		float *r[3]={x,y,z};
		float Displ[3]={DisplX, DisplY, DisplZ};
		int GridSize[3]={GS_X, GS_Y, GS_Z};
		int GS_XY=GS_X*GS_Y;
		float Rsm=R[i]+Rsmoth;
		int iRsm=(int)(Rsm+1.0);
		float RleftSQ=Rsm-0.707106781f;
		float RrightSQ=Rsm+0.707106781f;
		RleftSQ*=RleftSQ;
		RrightSQ*=RrightSQ;
		float RSQ=R[i]*R[i];
		float RsmSQ=Rsm*Rsm;
		
		int k;
		int start[3];
		int end[3];
		float rf[3];
		int rint[3];
		int ix,iy,iz;
		int gridpoint;
		
		float Rtmp;
		float ftmp;
		float RtmpSQ;
		float *vtmp;
		
		
		for(k=0;k<3;k++)
		{
			rf[k]=r[k][i]+Displ[k];
			rint[k]=(int)(rf[k]+0.5);
			start[k]=rint[k]-iRsm;
			end[k]=rint[k]+iRsm;
			if(start[k]<0)start[k]=0;
			if(end[k]>GridSize[k]-1)start[k]=GridSize[k]-1;
		}
		
		for(ix=start[0];ix<=end[0];ix++)
			for(iy=start[1];iy<=end[1];iy++)
				for(iz=start[2];iz<=end[2];iz++)
		{
			gridpoint=ix+iy*GS_X+iz*GS_XY;
			RtmpSQ=(rf[0]-ix)*(rf[0]-ix)+(rf[1]-iy)*(rf[1]-iy)+(rf[2]-iz)*(rf[2]-iz);
			vtmp=Surf+4*gridpoint;
			if(RtmpSQ<=RSQ)//RtmpSQ<=RSQ
			{
				Field[gridpoint]=iValue;
				vtmp[0]=-BIGDISTANSE;
				vtmp[3]=0.0;
			}
			else//RSQ<RtmpSQ
			{
				Rtmp=sqrt(RtmpSQ);
				if(RtmpSQ<=RsmSQ)//RSQ<RtmpSQ<RleftSQ
				{
					if(Field[gridpoint]==iBulkValue)
					{
						vtmp[0]=-BIGDISTANSE;
						Field[gridpoint]=-iValue;
						vtmp[3]=Rtmp-R[i];
					}
					else if(Field[gridpoint]<0&&Rtmp-R[i]<vtmp[3])
					{
						vtmp[0]=-BIGDISTANSE;
						Field[gridpoint]=-iValue;
						vtmp[3]=Rtmp-R[i];
					}
				}
				if(RleftSQ<=RtmpSQ&&RtmpSQ<=RrightSQ)//RleftSQ<=RtmpSQ<=RrightSQ
				{
					if(vtmp[0]>-100.0f)//e.i. is intersection
					{
						if(Rtmp-R[i]<vtmp[3])
						{
							ftmp=Rsm/Rtmp;
							vtmp[0]=rf[0]+ftmp*(ix-rf[0]);
							vtmp[1]=rf[1]+ftmp*(iy-rf[1]);
							vtmp[2]=rf[2]+ftmp*(iz-rf[2]);
							vtmp[3]=Rtmp-R[i];
						}
					}
					else if(Field[gridpoint]==iBulkValue)
					{
						ftmp=Rsm/Rtmp;
						vtmp[0]=rf[0]+ftmp*(ix-rf[0]);
						vtmp[1]=rf[1]+ftmp*(iy-rf[1]);
						vtmp[2]=rf[2]+ftmp*(iz-rf[2]);
						vtmp[3]=Rtmp-R[i];
					}
				}
			}
		}
	}
}
extern "C" int BuildAtomsDielPreMapsOnCUDA(GOAtomsStruct* atms,float *Displ)
{
	int iValue=atms->iDiel;
	int iBulkValue=atms->iDielBulk;
	float Rsmoth=atms->Rsmoth;
	int Natoms=atms->Natoms;
	float *r[3]={atms->r[0],atms->r[1],atms->r[2]};
	float *R=atms->R;
	float *Surf=atms->Surf;
	int *Field=atms->iVtmp;
	DbgPrint2("GOAtoms::BuildPreMaps(iValue=%d iBulkValue=%d)\n"
			,iValue,iBulkValue);
// 	DbgPrint2("\t Rion=%f[grids] Rsmoth=%f[grids] \nDispl=[%f,%f,%f][grids,grids,grids]\n", Rion, Rsmoth, Displ[0], Displ[1], Displ[2]);
// 	DbgPrint2("\t r[0]=[%f,%f,%f] R=%f q=%f Natom=%d\n"
// 			,r[0][0],r[1][0],r[2][0],R[0],q[0],NAtoms);
// 	DbgPrint2("\t r[0]=[%f,%f,%f] R=%f q=%f Natom=%d\n"
// 			,r[0][NAtoms-1],r[1][NAtoms-1],r[2][NAtoms-1],R[NAtoms-1],q[NAtoms-1],NAtoms);
	int i,j,k,gridpoint,rint[3];
	float RSQ,RsmSQ,RtmpSQ,Rsm,Rtmp;
	float RleftSQ,RrightSQ;
	float rf[3];
	int iRsm;
	int ix,iy,iz;
	float ftmp;
	float *vtmp;
	
	int GridSize[3]={atms->GS_X,atms->GS_Y,atms->GS_Z};
	int GSX=GridSize[0];
	int GSXY=GridSize[0]*GridSize[1];
	int GSXYZ=GridSize[0]*GridSize[1]*GridSize[2];
	
	int start[3];
	int end[3];
	
	int DoGPU=1;
	if(DoGPU)
	{
		int BlockSize=256;
		int GridOfBlocksSize=Natoms/BlockSize;
		if(Natoms%BlockSize!=0)
			GridOfBlocksSize++;
		
		float *cuda_r[3];
		float *cuda_R;
		float *cuda_Surf;
		int *cuda_Field;
		
		hipMalloc((void**)&cuda_r[0], Natoms*sizeof(float));
		hipMalloc((void**)&cuda_r[1], Natoms*sizeof(float));
		hipMalloc((void**)&cuda_r[2], Natoms*sizeof(float));
		
		hipMalloc((void**)&cuda_R, GSXYZ*sizeof(float));
		
		hipMalloc((void**)&cuda_Field, GSXYZ*sizeof(int));
		hipMalloc((void**)&cuda_Surf, GSXYZ*4*sizeof(float));
			
		hipMemcpy(cuda_r[0], r[0], Natoms*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(cuda_r[1], r[1], Natoms*sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(cuda_r[2], r[2], Natoms*sizeof(float), hipMemcpyHostToDevice);
		
		hipMemcpy(cuda_R, R, Natoms*sizeof(float), hipMemcpyHostToDevice);
		
		hipMemcpy(cuda_Field, Field, GSXYZ*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(cuda_Surf, Surf, GSXYZ*4*sizeof(float), hipMemcpyHostToDevice);
		
		KBuildAtomsDielPreMaps<<<GridOfBlocksSize, BlockSize>>>(cuda_r[0],cuda_r[1],cuda_r[2],cuda_R,cuda_Surf,cuda_Field, atms->GS_X,atms->GS_Y,atms->GS_Z,Rsmoth, Displ[0], Displ[1], Displ[2],iValue, iBulkValue, Natoms);
		hipDeviceSynchronize();
		
		hipMemcpy(Field, cuda_Field, GSXYZ*sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(Surf, cuda_Surf, GSXYZ*4*sizeof(float), hipMemcpyDeviceToHost);
		
		hipFree(cuda_Surf);
		hipFree(cuda_Field);
		hipFree(cuda_R);
		hipFree(cuda_r[2]);
		hipFree(cuda_r[1]);
		hipFree(cuda_r[0]);
		
		printf("GridSize [%d %d %d]\n",GridSize[0],GridSize[1],GridSize[2]);
		printf("\tGridSize [%d %d %d]\n",Field[0],Field[1],Field[2]);
		printf("r [%f %f %f]\n",r[0][0],r[1][0],r[2][0]);
		printf("\tr [%f %f %f]\n",Surf[0],Surf[1],Surf[2]);
	}
	else
	{
		for(i=0;i<Natoms;i++)
		{
			Rsm=R[i]+Rsmoth;
			iRsm=(int)(Rsm+1.0);
			RleftSQ=Rsm-0.707106781f;
			RrightSQ=Rsm+0.707106781f;
			RleftSQ*=RleftSQ;
			RrightSQ*=RrightSQ;
			RSQ=R[i]*R[i];
			RsmSQ=Rsm*Rsm;
			
			for(k=0;k<3;k++)
			{
				rf[k]=r[k][i]+Displ[k];
				rint[k]=(int)(rf[k]+0.5);
				start[k]=rint[k]-iRsm;
				end[k]=rint[k]+iRsm;
				if(start[k]<0)start[k]=0;
				if(end[k]>GridSize[k]-1)start[k]=GridSize[k]-1;
			}
			for(ix=start[0];ix<=end[0];ix++)
				for(iy=start[1];iy<=end[1];iy++)
					for(iz=start[2];iz<=end[2];iz++)
			{
				gridpoint=ix+iy*GSX+iz*GSXY;
				RtmpSQ=(rf[0]-ix)*(rf[0]-ix)+(rf[1]-iy)*(rf[1]-iy)+(rf[2]-iz)*(rf[2]-iz);
				vtmp=Surf+4*gridpoint;
				if(RtmpSQ<=RSQ)//RtmpSQ<=RSQ
				{
					Field[gridpoint]=iValue;
					vtmp[0]=-BIGDISTANSE;
					vtmp[3]=0.0;
				}
				else//RSQ<RtmpSQ
				{
					Rtmp=sqrt(RtmpSQ);
					if(RtmpSQ<=RsmSQ)//RSQ<RtmpSQ<RleftSQ
					{
						if(Field[gridpoint]==iBulkValue)
						{
							vtmp[0]=-BIGDISTANSE;
							Field[gridpoint]=-iValue;
							vtmp[3]=Rtmp-R[i];
						}
						else if(Field[gridpoint]<0&&Rtmp-R[i]<vtmp[3])
						{
							vtmp[0]=-BIGDISTANSE;
							Field[gridpoint]=-iValue;
							vtmp[3]=Rtmp-R[i];
						}
					}
					if(RleftSQ<=RtmpSQ&&RtmpSQ<=RrightSQ)//RleftSQ<=RtmpSQ<=RrightSQ
					{
						if(vtmp[0]>-100.0f)//e.i. is intersection
						{
							if(Rtmp-R[i]<vtmp[3])
							{
								ftmp=Rsm/Rtmp;
								vtmp[0]=rf[0]+ftmp*(ix-rf[0]);
								vtmp[1]=rf[1]+ftmp*(iy-rf[1]);
								vtmp[2]=rf[2]+ftmp*(iz-rf[2]);
								vtmp[3]=Rtmp-R[i];
							}
						}
						else if(Field[gridpoint]==iBulkValue)
						{
							ftmp=Rsm/Rtmp;
							vtmp[0]=rf[0]+ftmp*(ix-rf[0]);
							vtmp[1]=rf[1]+ftmp*(iy-rf[1]);
							vtmp[2]=rf[2]+ftmp*(iz-rf[2]);
							vtmp[3]=Rtmp-R[i];
						}
					}
				}
			}
		}
	}
	return EXIT_SUCCESS;
}
__global__ void KFinalazeSEVOnCUDA(float3 *surf_points,int *Field, float Rsmooth, int iBulkValue,int4 GS,int Nsurf_points)
{
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	int ix,iy,iz,gridpoint;
	float RsmSQ,RtmpSQ;
	
	int3 rint;
	int3 start;
	int3 end;
	
	if(i<Nsurf_points)
	{
		float RsmSQ=Rsmooth*Rsmooth;
		int lim=(int)(Rsmooth+0.5);
		
		rint.x=(int)(surf_points[i].x+0.5);
		rint.y=(int)(surf_points[i].y+0.5);
		rint.z=(int)(surf_points[i].z+0.5);
		
		start.x=rint.x-lim;
		start.y=rint.y-lim;
		start.z=rint.z-lim;
		
		end.x=rint.x+lim;
		end.y=rint.y+lim;
		end.z=rint.z+lim;
		
		if(start.x<0)start.x=0;
		if(start.y<0)start.y=0;
		if(start.z<0)start.z=0;
		
		if(end.x>GS.x-1)end.x=GS.x-1;
		if(end.y>GS.y-1)end.x=GS.y-1;
		if(end.z>GS.z-1)end.x=GS.z-1;
		
		for(ix=start.x;ix<=end.x;ix++)
			for(iy=start.y;iy<=end.y;iy++)
				for(iz=start.z;iz<=end.z;iz++)
		{
			gridpoint=ix+iy*GS.x+iz*GS.w;
			RtmpSQ=(surf_points[i].x-ix)*(surf_points[i].x-ix)+(surf_points[i].y-iy)*(surf_points[i].y-iy)+(surf_points[i].z-iz)*(surf_points[i].z-iz);
			if(RtmpSQ<RsmSQ&&Field[gridpoint]<0)
			{
				Field[gridpoint]=iBulkValue;
			}
		}
	}
}
extern "C" int FinalazeSEVOnCUDA2(int *GridSize,int *Field,int iBulkValue,float Rsmooth,float3 *surf_points, int Nsurf_points)
{
	int i;
	int GS_XYZ=GridSize[0]*GridSize[1]*GridSize[2];
	int itmp=GS_XYZ*4;
	
	int BlockSize=512;
	int GridOfBlocksSize=Nsurf_points/BlockSize;
	if(GridOfBlocksSize>65535)
		printf("Error: GridOfBlocksSize>65535\n");
	if(GS_XYZ%BlockSize!=0)
		GridOfBlocksSize++;
		
	printf("BlockSize=%d GridOfBlocksSize=%d\n",BlockSize,GridOfBlocksSize);
		
	float3 *cuda_surf_points;
	int *cuda_Field;
			
	hipMalloc((void**)&cuda_Field, GS_XYZ*sizeof(int));
	hipMalloc((void**)&cuda_surf_points, Nsurf_points*sizeof(float3));
				
	hipMemcpy(cuda_Field, Field, GS_XYZ*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(cuda_surf_points, surf_points, Nsurf_points*sizeof(float3), hipMemcpyHostToDevice);
		
	int4 GS={GridSize[0],GridSize[1],GridSize[2],GridSize[0]*GridSize[1]};
		
	KFinalazeSEVOnCUDA<<<GridOfBlocksSize, BlockSize>>>(cuda_surf_points,cuda_Field,Rsmooth, iBulkValue,GS,Nsurf_points);
	hipDeviceSynchronize();
			
	hipMemcpy(Field, cuda_Field, GS_XYZ*sizeof(int), hipMemcpyDeviceToHost);
			
	hipFree(cuda_surf_points);
	hipFree(cuda_Field);
	checkCUDAError("FinalazeSEVOnCUDA");
}
extern "C" int FinalazeSEVOnCUDA(int *GridSize,int *Field,int iBulkValue,float Rsmooth,float *Surf)
{
	DbgPrint2("BuildSES iBulkValue=%d\n",iBulkValue);
	int DoGPU=1;
	if(DoGPU)
	{
		int i;
		int Nsurf_points=0;
		int GS_XYZ=GridSize[0]*GridSize[1]*GridSize[2];
		int itmp=GS_XYZ*4;
		
		for(i=0;i<itmp;i=i+4)
			if(Surf[i]>-100.0)
				Nsurf_points++;
		
		float3* surf_points=new float3[Nsurf_points];
		printf("Nsurf_points=%d\n",Nsurf_points);
		
		int count=0;
		for(i=0;i<itmp;i=i+4)
			if(Surf[i]>-100.0)
		{
			surf_points[count].x=Surf[i];
			surf_points[count].y=Surf[i+1];
			surf_points[count].z=Surf[i+2];
			count++;
		}
		
		int BlockSize=512;
		int GridOfBlocksSize=Nsurf_points/BlockSize;
		if(GridOfBlocksSize>65535)
			printf("Error: GridOfBlocksSize>65535\n");
		if(GS_XYZ%BlockSize!=0)
			GridOfBlocksSize++;
		
		printf("BlockSize=%d GridOfBlocksSize=%d\n",BlockSize,GridOfBlocksSize);
		
		float3 *cuda_surf_points;
		int *cuda_Field;
			
		hipMalloc((void**)&cuda_Field, GS_XYZ*sizeof(int));
		hipMalloc((void**)&cuda_surf_points, Nsurf_points*sizeof(float3));
				
		hipMemcpy(cuda_Field, Field, GS_XYZ*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(cuda_surf_points, surf_points, Nsurf_points*sizeof(float3), hipMemcpyHostToDevice);
		
		int4 GS={GridSize[0],GridSize[1],GridSize[2],GridSize[0]*GridSize[1]};
		
		KFinalazeSEVOnCUDA<<<GridOfBlocksSize, BlockSize>>>(cuda_surf_points,cuda_Field,Rsmooth, iBulkValue,GS,Nsurf_points);
		hipDeviceSynchronize();
			
		hipMemcpy(Field, cuda_Field, GS_XYZ*sizeof(int), hipMemcpyDeviceToHost);
			
		hipFree(cuda_surf_points);
		hipFree(cuda_Field);
		delete [] surf_points;
		checkCUDAError("FinalazeSEVOnCUDA");
		
		for(i=0;i<GS_XYZ;i++)
		{
			if(Field[i]<0)
			{
				Field[i]=-Field[i];
			}
		}
	}
	else
	{
		int i,j,k,ix,iy,iz,gridpoint,rint[3];
		float RsmSQ,RtmpSQ;
		int GS_X=GridSize[0];
		int GS_XY=GridSize[0]*GridSize[1];
		int GS_XYZ=GridSize[0]*GridSize[1]*GridSize[2];
		int start[3];
		int end[3];
		int lim=(int)(Rsmooth+0.5);
		int itmp=GS_XYZ*4;
		float *vtmp;
		
		RsmSQ=Rsmooth*Rsmooth;
		for(i=0;i<itmp;i=i+4)
		{
			if(Surf[i]>-100.0)
			{
				//DbgPrint2("BuildSES %d\n",i/4);
				vtmp=Surf+i;
				for(k=0;k<3;k++)
				{
					rint[k]=(int)(vtmp[k]+0.5);
					start[k]=rint[k]-lim;
					end[k]=rint[k]+lim;
					if(start[k]<0)start[k]=0;
					if(end[k]>GridSize[k]-1)end[k]=GridSize[k]-1;
				}
				for(ix=start[0];ix<=end[0];ix++)
					for(iy=start[1];iy<=end[1];iy++)
						for(iz=start[2];iz<=end[2];iz++)
				{
					gridpoint=ix+iy*GS_X+iz*GS_XY;
					RtmpSQ=(vtmp[0]-ix)*(vtmp[0]-ix)+(vtmp[1]-iy)*(vtmp[1]-iy)+(vtmp[2]-iz)*(vtmp[2]-iz);
					if(RtmpSQ<RsmSQ&&Field[gridpoint]<0)
					{
						Field[gridpoint]=iBulkValue;
					}
				}
			}
		}
		for(i=0;i<GS_XYZ;i++)
		{
			if(Field[i]<0)
			{
				Field[i]=-Field[i];
			}
		}
	}
	//RemovingCavitiesOnDielectricMap
// 	if(RemovingCavitiesOnDielectricMap)
// 	{
// 		DbgPrint2("RemovingCavitiesOnDielectricMap\n");
// 		RemovingCavitiesAtValues(GS_X,GS_Y,GS_Z,Field, RemCavOnDielWhere2Look ,RemCavOnDielFillWith);
// 	}
	return EXIT_SUCCESS;
}
__global__ void KGOAtomsCoulBCXYOnCUDA(float4 *rq,int Nq, int4 GS,float eps,float *Potential)
{
	int i=blockIdx.x*blockDim.x+threadIdx.x;
	int j=blockIdx.y;
	int iq;
	int k,GrPnt;
	float r1;
	
	
	if(i<GS.x && j<GS.y)
	{
		k=0;
		GrPnt=i+j*GS.x+k*GS.w;
					
		for(iq=0;iq < Nq;iq++)
		{
			r1=sqrt((i-rq[iq].x)*(i-rq[iq].x) + (j-rq[iq].y)*(j-rq[iq].y) + (k-rq[iq].z)*(k-rq[iq].z));
			Potential[GrPnt]+=rq[iq].w/(eps*r1);
		}
					
		k=GS.z-1;
		GrPnt=i+j*GS.x+k*GS.w;
					
		for(iq=0;iq < Nq;iq++)
		{
			r1=sqrt((i-rq[iq].x)*(i-rq[iq].x) + (j-rq[iq].y)*(j-rq[iq].y) + (k-rq[iq].z)*(k-rq[iq].z));
			Potential[GrPnt]+=rq[iq].w/(eps*r1);
		}
	}
}
__global__ void KGOAtomsCoulBCYZOnCUDA(float4 *rq,int Nq, int4 GS,float eps,float *Potential, char3 cBldBCatPlane)
{
	int j=blockIdx.x*blockDim.x+threadIdx.x;
	int k=blockIdx.y+cBldBCatPlane.z;
	int iq;
	int i,GrPnt;
	float r1;
	
	if(j<GS.y && k<GS.z)
	{
		i=0;
		GrPnt=i+j*GS.x+k*GS.w;
					
		for(iq=0;iq < Nq;iq++)
		{
			r1=sqrt((i-rq[iq].x)*(i-rq[iq].x) + (j-rq[iq].y)*(j-rq[iq].y) + (k-rq[iq].z)*(k-rq[iq].z));
			Potential[GrPnt]+=rq[iq].w/(eps*r1);
		}
					
		i=GS.x-1;
		GrPnt=i+j*GS.x+k*GS.w;
					
		for(iq=0;iq < Nq;iq++)
		{
			r1=sqrt((i-rq[iq].x)*(i-rq[iq].x) + (j-rq[iq].y)*(j-rq[iq].y) + (k-rq[iq].z)*(k-rq[iq].z));
			Potential[GrPnt]+=rq[iq].w/(eps*r1);
		}
	}
}
__global__ void KGOAtomsCoulBCXZOnCUDA(float4 *rq,int Nq, int4 GS,float eps,float *Potential, char3 cBldBCatPlane)
{
	int i=blockIdx.x*blockDim.x+threadIdx.x+cBldBCatPlane.x;
	int j;
	int k=blockIdx.y+cBldBCatPlane.z;
	int iq;
	int GrPnt;
	float r1;
	if(i<GS.x && k<GS.z)
	{
		j=0;
		GrPnt=i+j*GS.x+k*GS.w;
					
		for(iq=0;iq < Nq;iq++)
		{
			r1=sqrt((i-rq[iq].x)*(i-rq[iq].x) + (j-rq[iq].y)*(j-rq[iq].y) + (k-rq[iq].z)*(k-rq[iq].z));
			Potential[GrPnt]+=rq[iq].w/(eps*r1);
		}
					
		j=GS.y-1;
		GrPnt=i+j*GS.x+k*GS.w;
					
		for(iq=0;iq < Nq;iq++)
		{
			r1=sqrt((i-rq[iq].x)*(i-rq[iq].x) + (j-rq[iq].y)*(j-rq[iq].y) + (k-rq[iq].z)*(k-rq[iq].z));
			Potential[GrPnt]+=rq[iq].w/(eps*r1);
		}
	}
}
extern "C" int GOAtoms_SetCoulombicBC(int *GridSize,int *BldBCatPlane,float4* rq,int Nq,float eps,float *Potential)
{
	int DoGPU=1;
	if(DoGPU)
	{
		int4 GS;
		GS.x=GridSize[0];
		GS.y=GridSize[1];
		GS.z=GridSize[2];
		GS.w=GridSize[0]*GridSize[1];
		
		float4 *cuda_rq;
		
		hipMalloc((void**)&cuda_rq, Nq*sizeof(float4));
		
		
		hipMemcpy(cuda_rq, rq, Nq*sizeof(float4), hipMemcpyHostToDevice);
		
		dim3 dimBlock(512,1,1);
		dim3 dimGrid(1,1,1);
		char3 cBldBCatPlane;
		
		cBldBCatPlane.x=BldBCatPlane[0];
		cBldBCatPlane.y=BldBCatPlane[1];
		cBldBCatPlane.z=BldBCatPlane[2];
		
		if(BldBCatPlane[2])
		{
			float *cuda_pot;
			hipMalloc((void**)&cuda_pot, GS.w*GS.z*sizeof(float));
			hipMemcpy(cuda_pot, Potential, GS.w*GS.z*sizeof(float), hipMemcpyHostToDevice);
			
			if(GS.x<=512)
			{
				dimBlock.x=GS.x;
				dimGrid.x=1;
			}
			else
			{
				dimBlock.x=512;
				dimGrid.x=GS.x/512;
				if(GS.x%512!=0)dimGrid.x++;
			}
			dimGrid.y=GS.y;
			
			printf("dimBlock [%d,%d,%d]\n",dimBlock.x,dimBlock.y,dimBlock.z);
			printf("dimGrid [%d,%d,%d]\n",dimGrid.x,dimGrid.y,dimGrid.z);
			KGOAtomsCoulBCXYOnCUDA<<<dimGrid, dimBlock>>>(cuda_rq,Nq, GS,eps, cuda_pot);
			hipDeviceSynchronize();
			
			hipMemcpy(Potential, cuda_pot, GS.w*GS.z*sizeof(float), hipMemcpyDeviceToHost);
			
			hipFree(cuda_pot);
		}
		if(BldBCatPlane[0])
		{
			float *cuda_pot;
			hipMalloc((void**)&cuda_pot, GS.w*GS.z*sizeof(float));
			hipMemcpy(cuda_pot, Potential, GS.w*GS.z*sizeof(float), hipMemcpyHostToDevice);
			
			if(GS.y<=512)
			{
				dimBlock.x=GS.y;
				dimGrid.x=1;
			}
			else
			{
				dimBlock.x=512;
				dimGrid.x=GS.y/512;
				if(GS.y%512!=0)dimGrid.x++;
			}
			dimGrid.y=GS.z;
			if(BldBCatPlane[2])
			{
				dimGrid.y=GS.z-2;
			}
			printf("dimBlock [%d,%d,%d]\n",dimBlock.x,dimBlock.y,dimBlock.z);
			printf("dimGrid [%d,%d,%d]\n",dimGrid.x,dimGrid.y,dimGrid.z);
			KGOAtomsCoulBCYZOnCUDA<<<dimGrid, dimBlock>>>(cuda_rq,Nq, GS,eps, cuda_pot,cBldBCatPlane);
			hipDeviceSynchronize();
			
			hipMemcpy(Potential, cuda_pot, GS.w*GS.z*sizeof(float), hipMemcpyDeviceToHost);
			
			hipFree(cuda_pot);
		}
		if(BldBCatPlane[1])
		{
			float *cuda_pot;
			hipMalloc((void**)&cuda_pot, GS.w*GS.z*sizeof(float));
			hipMemcpy(cuda_pot, Potential, GS.w*GS.z*sizeof(float), hipMemcpyHostToDevice);
			
			if(GS.y<=512)
			{
				if(BldBCatPlane[0])
				{
					dimBlock.x=GS.x-2;
					dimGrid.x=1;
				}
				else
				{
					dimBlock.x=GS.x;
					dimGrid.x=1;
				}
			}
			else
			{
				if(BldBCatPlane[0])
				{
					dimBlock.x=512;
					dimGrid.x=(GS.x-2)/512;
					if((GS.x-2)%512!=0)dimGrid.x++;
				}
				else
				{
					dimBlock.x=512;
					dimGrid.x=GS.x/512;
					if(GS.x%512!=0)dimGrid.x++;
				}
				
			}
			dimGrid.y=GS.z;
			if(BldBCatPlane[2])
			{
				dimGrid.y=GS.z-2;
			}
			printf("dimBlock [%d,%d,%d]\n",dimBlock.x,dimBlock.y,dimBlock.z);
			printf("dimGrid [%d,%d,%d]\n",dimGrid.x,dimGrid.y,dimGrid.z);
			KGOAtomsCoulBCXZOnCUDA<<<dimGrid, dimBlock>>>(cuda_rq,Nq, GS,eps, cuda_pot,cBldBCatPlane);
			hipDeviceSynchronize();
			
			hipMemcpy(Potential, cuda_pot, GS.w*GS.z*sizeof(float), hipMemcpyDeviceToHost);
			
			hipFree(cuda_pot);
		}
		hipFree(cuda_rq);
	}
	else
	{
		int4 GS;
		GS.x=GridSize[0];
		GS.y=GridSize[1];
		GS.z=GridSize[2];
		GS.w=GridSize[0]*GridSize[1];
		
		int GrPnt;
		int iq;
		int i,j,k;
		int iStart,jStart,kStart;
		int iEnd,jEnd,kEnd;
		
		float r1;
		
		if(BldBCatPlane[2])
		{
			for(i=0;i<GS.x;i++)
				for(j=0;j<GS.y;j++)//XY
			{
				k=0;
				GrPnt=i+j*GS.x+k*GS.w;
					
				for(iq=0;iq < Nq;iq++)
				{
					r1=sqrt((i-rq[iq].x)*(i-rq[iq].x) + (j-rq[iq].y)*(j-rq[iq].y) + (k-rq[iq].z)*(k-rq[iq].z));
					Potential[GrPnt]+=rq[iq].w/(eps*r1);
				}
					
				k=GS.z-1;
				GrPnt=i+j*GS.x+k*GS.w;
					
				for(iq=0;iq < Nq;iq++)
				{
					r1=sqrt((i-rq[iq].x)*(i-rq[iq].x) + (j-rq[iq].y)*(j-rq[iq].y) + (k-rq[iq].z)*(k-rq[iq].z));
					Potential[GrPnt]+=rq[iq].w/(eps*r1);
				}
			}
		}
		if(BldBCatPlane[0])
		{
			jStart=0;
			kStart=0;
			jEnd=GS.y;
			kEnd=GS.z;
			if(BldBCatPlane[2])
			{
				kStart=1;
				kEnd=GS.z-1;
			}
			for(j=jStart;j<jEnd;j++)//YZ
				for(k=kStart;k<kEnd;k++)
			{
				i=0;
				GrPnt=i+j*GS.x+k*GS.w;
					
				for(iq=0;iq < Nq;iq++)
				{
					r1=sqrt((i-rq[iq].x)*(i-rq[iq].x) + (j-rq[iq].y)*(j-rq[iq].y) + (k-rq[iq].z)*(k-rq[iq].z));
					Potential[GrPnt]+=rq[iq].w/(eps*r1);
				}
					
				i=GS.x-1;
				GrPnt=i+j*GS.x+k*GS.w;
					
				for(iq=0;iq < Nq;iq++)
				{
					r1=sqrt((i-rq[iq].x)*(i-rq[iq].x) + (j-rq[iq].y)*(j-rq[iq].y) + (k-rq[iq].z)*(k-rq[iq].z));
					Potential[GrPnt]+=rq[iq].w/(eps*r1);
				}
			}
		}
		if(BldBCatPlane[1])
		{
			iStart=0;
			kStart=0;
			iEnd=GS.x;
			kEnd=GS.z;
			if(BldBCatPlane[2])
			{
				kStart=1;
				kEnd=GS.z-1;
			}
			if(BldBCatPlane[0])
			{
				iStart=1;
				iEnd=GS.x-1;
			}
			for(i=iStart;i<iEnd;i++)//ZX
				for(k=kStart;k<kEnd;k++)
			{
				j=0;
				GrPnt=i+j*GS.x+k*GS.w;
					
				for(iq=0;iq < Nq;iq++)
				{
					r1=sqrt((i-rq[iq].x)*(i-rq[iq].x) + (j-rq[iq].y)*(j-rq[iq].y) + (k-rq[iq].z)*(k-rq[iq].z));
					Potential[GrPnt]+=rq[iq].w/(eps*r1);
				}
					
				j=GS.y-1;
				GrPnt=i+j*GS.x+k*GS.w;
					
				for(iq=0;iq < Nq;iq++)
				{
					r1=sqrt((i-rq[iq].x)*(i-rq[iq].x) + (j-rq[iq].y)*(j-rq[iq].y) + (k-rq[iq].z)*(k-rq[iq].z));
					Potential[GrPnt]+=rq[iq].w/(eps*r1);
				}
			}
		}
	}
	return EXIT_SUCCESS;
}

#endif
