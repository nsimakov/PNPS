#include "hip/hip_runtime.h"
#ifndef _CUDA_SOLVER_CU_
#define _CUDA_SOLVER_CU_

#include <stdio.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>

#include "pnpstructs.h"
//#include "pnpdebug.h"
#define DefClock0 clock_t time0;timeval tvtime0,tvtime1;
#define StartClock0 time0=clock ();gettimeofday(&tvtime0,NULL);
#define StopClock0 gettimeofday(&tvtime1,NULL);DbgPrint0("Time : %g s(CPU Time) %g  s(Wall Time)\n",((double)(clock ()-time0))/CLOCKS_PER_SEC,double(tvtime1.tv_sec)+(double(tvtime1.tv_usec)/1000000.0)-double(tvtime0.tv_sec)-(double(tvtime0.tv_usec)/1000000.0));
#define StopClockWMes0(Massege) gettimeofday(&tvtime1,NULL);printf("Time for %s is %.5g s (CPU Time) %g  s(Wall Time)\n",(Massege),((double)(clock ()-time0))/CLOCKS_PER_SEC,double(tvtime1.tv_sec)+(double(tvtime1.tv_usec)/1000000.0)-double(tvtime0.tv_sec)-(double(tvtime0.tv_usec)/1000000.0));

#define FMUL __fmul_rn
#define FADD __fadd_rn
#define FMAF __fmaf_rn

__constant__ float  dc_om1;
__constant__ float  dc_om2d6;
__constant__ float* dc_P[8];
__constant__ int dc_Qnum[8];
__constant__ float* dc_Q[8];
__constant__ float* dc_Qmult[8];
__constant__ int* dc_Qpos[8];

__constant__ int dc_DBnum[8];
__constant__ float* dc_DielMult[48];
__constant__ int* dc_DBpos[8];
#define dc_lookupVirGridSIZE 1024
__constant__ int  dc_lookupVirGrid[dc_lookupVirGridSIZE];


__global__ void KFLaplaceB(float* d_P0,float* d_P1,float* d_P2,float* d_P3,
																float* d_P4,float* d_P5,float* d_P6,float* d_P7,
																const float om1,const float om2d6,
								const int pitchX,const int pitchXY,const int pitchXY_BS_Z,const int BS_X,const int BS_XY)
{
// 	int VirXblock = (blockIdx.x%VirGridX);
// 	int VirYblock = (blockIdx.x/VirGridX);
// 	int VirZblock = blockIdx.y;
// 	
// 	int tx = VirXblock*BS_X + threadIdx.x;
// 	int ty = VirYblock*BS_Y + threadIdx.y+1;
// 	int tz = VirZblock*BS_Z + threadIdx.z+1;
// 	
// 	int i=tx+ty*pitchX+tz*pitchXY;
	int t=threadIdx.x+threadIdx.y*pitchX+threadIdx.z*pitchXY;
	int i=dc_lookupVirGrid[blockIdx.x] + blockIdx.y*pitchXY_BS_Z + t;
	t=threadIdx.x+threadIdx.y*BS_X+threadIdx.z*BS_XY;
	
	float xP0,xP3,xP5,xP6;
	float yP0,yP3,yP5,yP6;
	float zP0,zP3,zP5,zP6;
	
	__shared__ float shP[256];
	
	
	//do over P1
	shP[t]=d_P1[i];
	__syncthreads();
	
	zP5=FADD(shP[t],d_P1[i+pitchXY]);
	yP3=FADD(shP[t],d_P1[i+pitchX]);
	if(threadIdx.x!=0)
		xP0=FADD(shP[t],shP[t-1]);
	else
		xP0=FADD(shP[t],d_P1[i-1]);
	__syncthreads();
	//do over P2
	shP[t]=d_P2[i];
	__syncthreads();
	
	zP6=FADD(shP[t],d_P2[i+pitchXY]);
	yP0=FADD(shP[t],d_P2[i-pitchX]);
	if(threadIdx.x!=blockDim.x-1)
		xP3=FADD(shP[t],shP[t+1]);
	else
		xP3=FADD(shP[t],d_P2[i+1]);
	__syncthreads();
	//do over P4
	shP[t]=d_P4[i];
	__syncthreads();
	zP0=FADD(shP[t],d_P4[i-pitchXY]);
	yP6=FADD(shP[t],d_P4[i+pitchX]);
	if(threadIdx.x!=blockDim.x-1)
		xP5=FADD(shP[t],shP[t+1]);
	else
		xP5=FADD(shP[t],d_P4[i+1]);
	__syncthreads();
	//do over P7
	shP[t]=d_P7[i];
	__syncthreads();
	zP3=FADD(shP[t],d_P7[i-pitchXY]);
	yP5=FADD(shP[t],d_P7[i-pitchX]);
	if(threadIdx.x!=0)
		xP6=FADD(shP[t],shP[t-1]);
	else
		xP6=FADD(shP[t],d_P7[i-1]);
	
	
	d_P0[i]=FADD(FMUL(om1,d_P0[i]),FMUL(om2d6,FADD(FADD(xP0,yP0),zP0)));
	d_P3[i]=FADD(FMUL(om1,d_P3[i]),FMUL(om2d6,FADD(FADD(xP3,yP3),zP3)));
	d_P5[i]=FADD(FMUL(om1,d_P5[i]),FMUL(om2d6,FADD(FADD(xP5,yP5),zP5)));
	d_P6[i]=FADD(FMUL(om1,d_P6[i]),FMUL(om2d6,FADD(FADD(xP6,yP6),zP6)));
}
__global__ void KFLaplaceW(float* d_P0,float* d_P1,float* d_P2,float* d_P3,
																float* d_P4,float* d_P5,float* d_P6,float* d_P7,
								const float om1,const float om2d6,
				const int pitchX,const int pitchXY,const int pitchXY_BS_Z,const int BS_X,const int BS_XY)
{
// 	int VirXblock = (blockIdx.x%VirGridX);
// 	int VirYblock = (blockIdx.x/VirGridX);
// 	int VirZblock = blockIdx.y;
// 	
// 	int tx = VirXblock*BS_X + threadIdx.x;
// 	int ty = VirYblock*BS_Y + threadIdx.y+1;
// 	int tz = VirZblock*BS_Z + threadIdx.z+1;
// 	
// 	int i=tx+ty*pitchX+tz*pitchXY;
	//int i=dc_lookupVirGrid[blockIdx.x] + blockIdx.y*pitchXY_BS_Z + threadIdx.x+threadIdx.y*pitchX+threadIdx.z*pitchXY;
	
	int t=threadIdx.x+threadIdx.y*pitchX+threadIdx.z*pitchXY;
	int i=dc_lookupVirGrid[blockIdx.x] + blockIdx.y*pitchXY_BS_Z + t;
	t=threadIdx.x+threadIdx.y*BS_X+threadIdx.z*BS_XY;
	
	float xP1,xP2,xP4,xP7;
	float yP1,yP2,yP4,yP7;
	float zP1,zP2,zP4,zP7;
	
	__shared__ float shP[256];
	
	//do dc_P[0]
	shP[t]=d_P0[i];
	__syncthreads();
	zP4=FADD(shP[t],d_P0[i+pitchXY]);
	yP2=FADD(shP[t],d_P0[i+pitchX]);
	if(threadIdx.x!=blockDim.x-1)
		xP1=FADD(shP[t],shP[t+1]);
	else
		xP1=FADD(shP[t],d_P0[i+1]);
	__syncthreads();
	//do d_P[3]
	shP[t]=d_P3[i];
	__syncthreads();
	zP7=FADD(shP[t],d_P3[i+pitchXY]);
	yP1=FADD(shP[t],d_P3[i-pitchX]);
	if(threadIdx.x!=0)
		xP2=FADD(shP[t],shP[t-1]);
	else
		xP2=FADD(shP[t],d_P3[i-1]);
	__syncthreads();
	//do d_P[5]
	shP[t]=d_P5[i];
	__syncthreads();
	zP1=FADD(shP[t],d_P5[i-pitchXY]);
	yP7=FADD(shP[t],d_P5[i+pitchX]);
	if(threadIdx.x!=0)
		xP4=FADD(shP[t],shP[t-1]);
	else
		xP4=FADD(shP[t],d_P5[i-1]);
	__syncthreads();
	//do d_P6
	shP[t]=d_P6[i];
	__syncthreads();
	zP2=FADD(shP[t],d_P6[i-pitchXY]);
	yP4=FADD(shP[t],d_P6[i-pitchX]);
	if(threadIdx.x!=blockDim.x-1)
		xP7=FADD(shP[t],shP[t+1]);
	else
		xP7=FADD(shP[t],d_P6[i+1]);
	
	//d_P6[i]=FADD(FMUL(om1,d_P6[i]),FMUL(om2d6,FADD(FADD(xP6,yP6),zP6)));
	
	d_P1[i]=FADD(FMUL(om1,d_P1[i]),FMUL(om2d6,FADD(FADD(xP1,yP1),zP1)));
	d_P2[i]=FADD(FMUL(om1,d_P2[i]),FMUL(om2d6,FADD(FADD(xP2,yP2),zP2)));
	d_P4[i]=FADD(FMUL(om1,d_P4[i]),FMUL(om2d6,FADD(FADD(xP4,yP4),zP4)));
	d_P7[i]=FADD(FMUL(om1,d_P7[i]),FMUL(om2d6,FADD(FADD(xP7,yP7),zP7)));
}
__global__ void KFPoissonQB()
{
	int t=threadIdx.x+blockIdx.x*blockDim.x;
	int i;
	if(t<dc_Qnum[0])
	{
		i=dc_Qpos[0][t];
		dc_P[0][i]=FADD(dc_P[0][i],dc_Q[0][t]);
	}
	if(t<dc_Qnum[3])
	{
		i=dc_Qpos[3][t];
		dc_P[3][i]=FADD(dc_P[3][i],dc_Q[3][t]);
	}
	if(t<dc_Qnum[5])
	{
		i=dc_Qpos[5][t];
		dc_P[5][i]=FADD(dc_P[5][i],dc_Q[5][t]);
	}
	if(t<dc_Qnum[6])
	{
		i=dc_Qpos[6][t];
		dc_P[6][i]=FADD(dc_P[6][i],dc_Q[6][t]);
	}
}
__global__ void KFPoissonQW()
{
	int t=threadIdx.x+blockIdx.x*blockDim.x;
	int i;
	if(t<dc_Qnum[1])
	{
		i=dc_Qpos[1][t];
		dc_P[1][i]=FADD(dc_P[1][i],dc_Q[1][t]);
	}
	if(t<dc_Qnum[2])
	{
		i=dc_Qpos[2][t];
		dc_P[2][i]=FADD(dc_P[2][i],dc_Q[2][t]);
	}
	if(t<dc_Qnum[4])
	{
		i=dc_Qpos[4][t];
		dc_P[4][i]=FADD(dc_P[4][i],dc_Q[4][t]);
	}
	if(t<dc_Qnum[7])
	{
		i=dc_Qpos[7][t];
		dc_P[7][i]=FADD(dc_P[7][i],dc_Q[7][t]);
	}
}
__global__ void KFPoissonDBB(float* d_P0,float* d_P1,float* d_P2,float* d_P3,
																float* d_P4,float* d_P5,float* d_P6,float* d_P7,
								const float om2d6,
				const int pitchX,const int pitchXY)
{
	int t=threadIdx.x+blockIdx.x*blockDim.x;
	int i;
	float xP,xM,yP,yM,zP,zM;
	if(t<dc_DBnum[0])
	{
		i=dc_DBpos[0][t];
		//P0 x
		xP=FMUL(dc_DielMult[PlusX][t],d_P1[i]);
		xM=FMUL(dc_DielMult[MinusX][t],d_P1[i-1]);
		//P0 y 
		yP=FMUL(dc_DielMult[PlusY][t],d_P2[i]);
		yM=FMUL(dc_DielMult[MinusY][t],d_P2[i-pitchX]);
		//P0 z
		zP=FMUL(dc_DielMult[PlusZ][t],d_P4[i]);
		zM=FMUL(dc_DielMult[MinusZ][t],d_P4[i-pitchXY]);
		
		xP=FADD(xM,xP);
		yP=FADD(yM,yP);
		zP=FADD(zM,zP);
		dc_P[0][i]=FADD(FADD(xP,yP),FADD(zP,dc_P[0][i]));
	}
	if(t<dc_DBnum[3])
	{
		i=dc_DBpos[3][t];
		//P3 x
		xM=FMUL(dc_DielMult[3*6+MinusX][t],d_P2[i]);
		xP=FMUL(dc_DielMult[3*6+PlusX][t],d_P2[i+1]);
		//P3 y
		yM=FMUL(dc_DielMult[3*6+MinusY][t],d_P1[i]);
		yP=FMUL(dc_DielMult[3*6+PlusY][t],d_P1[i+pitchX]);
		//P3 z
		zP=FMUL(dc_DielMult[3*6+PlusZ][t],d_P7[i]);
		zM=FMUL(dc_DielMult[3*6+MinusZ][t],d_P7[i-pitchXY]);
		
		xP=FADD(xM,xP);
		yP=FADD(yM,yP);
		zP=FADD(zM,zP);
		dc_P[3][i]=FADD(FADD(xP,yP),FADD(zP,dc_P[3][i]));
	}
	if(t<dc_DBnum[5])
	{
		i=dc_DBpos[5][t];
		//P5 x
		xM=FMUL(dc_DielMult[5*6+MinusX][t],d_P4[i]);
		xP=FMUL(dc_DielMult[5*6+PlusX][t],d_P4[i+1]);
		//P5 y
		yP=FMUL(dc_DielMult[5*6+PlusY][t],d_P7[i]);
		yM=FMUL(dc_DielMult[5*6+MinusY][t],d_P7[i-pitchX]);
		//P5 z
		zM=FMUL(dc_DielMult[5*6+MinusZ][t],d_P1[i]);
		zP=FMUL(dc_DielMult[5*6+PlusZ][t],d_P1[i+pitchXY]);
		
		xP=FADD(xM,xP);
		yP=FADD(yM,yP);
		zP=FADD(zM,zP);
		dc_P[5][i]=FADD(FADD(xP,yP),FADD(zP,dc_P[5][i]));
	}
	if(t<dc_DBnum[6])
	{
		i=dc_DBpos[6][t];
			//P6 x
		xP=FMUL(dc_DielMult[6*6+PlusX][t],d_P7[i]);
		xM=FMUL(dc_DielMult[6*6+MinusX][t],d_P7[i-1]);
		//P6 y
		yM=FMUL(dc_DielMult[6*6+MinusY][t],d_P4[i]);
		yP=FMUL(dc_DielMult[6*6+PlusY][t],d_P4[i+pitchX]);
		//P6 z
		zM=FMUL(dc_DielMult[6*6+MinusZ][t],d_P2[i]);
		zP=FMUL(dc_DielMult[6*6+PlusZ][t],d_P2[i+pitchXY]);
		
		xP=FADD(xM,xP);
		yP=FADD(yM,yP);
		zP=FADD(zM,zP);
		dc_P[6][i]=FADD(FADD(xP,yP),FADD(zP,dc_P[6][i]));
	}
}
__global__ void KFPoissonDBW(float* d_P0,float* d_P1,float* d_P2,float* d_P3,
																float* d_P4,float* d_P5,float* d_P6,float* d_P7,
								const float om2d6,
				const int pitchX,const int pitchXY)
{
	int t=threadIdx.x+blockIdx.x*blockDim.x;
	int i;
	float xP,xM,yP,yM,zP,zM;
	
	if(t<dc_DBnum[1])
	{
		i=dc_DBpos[1][t];
		//P1 x
		xM=FMUL(dc_DielMult[1*6+MinusX][t],d_P0[i]);
		xP=FMUL(dc_DielMult[1*6+PlusX][t],d_P0[i+1]);
		//P1 y
		yP=FMUL(dc_DielMult[1*6+PlusY][t],d_P3[i]);
		yM=FMUL(dc_DielMult[1*6+MinusY][t],d_P3[i-pitchX]);
		//P1 z
		zP=FMUL(dc_DielMult[1*6+PlusZ][t],d_P5[i]);
		zM=FMUL(dc_DielMult[1*6+MinusZ][t],d_P5[i-pitchXY]);
		
		xP=FADD(xM,xP);
		yP=FADD(yM,yP);
		zP=FADD(zM,zP);
		dc_P[1][i]=FADD(FADD(xP,yP),FADD(zP,dc_P[1][i]));
	}
	if(t<dc_DBnum[2])
	{
		i=dc_DBpos[2][t];
		//P2 x
		xP=FMUL(dc_DielMult[2*6+PlusX][t],d_P3[i]);
		xM=FMUL(dc_DielMult[2*6+MinusX][t],d_P3[i-1]);
		//P2 y
		yM=FMUL(dc_DielMult[2*6+MinusY][t],d_P0[i]);
		yP=FMUL(dc_DielMult[2*6+PlusY][t],d_P0[i+pitchX]);
		//P2 z
		zP=FMUL(dc_DielMult[2*6+PlusZ][t],d_P6[i]);
		zM=FMUL(dc_DielMult[2*6+MinusZ][t],d_P6[i-pitchXY]);
		
		xP=FADD(xM,xP);
		yP=FADD(yM,yP);
		zP=FADD(zM,zP);
		dc_P[2][i]=FADD(FADD(xP,yP),FADD(zP,dc_P[2][i]));
	}
	if(t<dc_DBnum[4])
	{
		i=dc_DBpos[4][t];
		//P4x
		xP=FMUL(dc_DielMult[4*6+PlusX][t],d_P5[i]);
		xM=FMUL(dc_DielMult[4*6+MinusX][t],d_P5[i-1]);
		//P4 y
		yP=FMUL(dc_DielMult[4*6+PlusY][t],d_P6[i]);
		yM=FMUL(dc_DielMult[4*6+MinusY][t],d_P6[i-pitchX]);
		//P4 z
		zM=FMUL(dc_DielMult[4*6+MinusZ][t],d_P0[i]);
		zP=FMUL(dc_DielMult[4*6+PlusZ][t],d_P0[i+pitchXY]);
		
		xP=FADD(xM,xP);
		yP=FADD(yM,yP);
		zP=FADD(zM,zP);
		dc_P[4][i]=FADD(FADD(xP,yP),FADD(zP,dc_P[4][i]));
	}
	if(t<dc_DBnum[7])
	{
		i=dc_DBpos[7][t];
		//P7 x
		xM=FMUL(dc_DielMult[7*6+MinusX][t],d_P6[i]);
		xP=FMUL(dc_DielMult[7*6+PlusX][t],d_P6[i+1]);
		//P7 y
		yM=FMUL(dc_DielMult[7*6+MinusY][t],d_P5[i]);
		yP=FMUL(dc_DielMult[7*6+PlusY][t],d_P5[i+pitchX]);
		//P7 z
		zM=FMUL(dc_DielMult[7*6+MinusZ][t],d_P3[i]);
		zP=FMUL(dc_DielMult[7*6+PlusZ][t],d_P3[i+pitchXY]);
		
		xP=FADD(xM,xP);
		yP=FADD(yM,yP);
		zP=FADD(zM,zP);
		dc_P[7][i]=FADD(FADD(xP,yP),FADD(zP,dc_P[7][i]));
	}
}

__global__ void KFPoissonQnCalcEB()
{
	int t=threadIdx.x+blockIdx.x*blockDim.x;
	int i;
	if(t<dc_Qnum[0])
	{
		i=dc_Qpos[0][t];
		dc_P[0][i]=FADD(dc_P[0][i],dc_Q[0][t]);
	}
	if(t<dc_Qnum[3])
	{
		i=dc_Qpos[3][t];
		dc_P[3][i]=FADD(dc_P[3][i],dc_Q[3][t]);
	}
	if(t<dc_Qnum[5])
	{
		i=dc_Qpos[5][t];
		dc_P[5][i]=FADD(dc_P[5][i],dc_Q[5][t]);
	}
	if(t<dc_Qnum[6])
	{
		i=dc_Qpos[6][t];
		dc_P[6][i]=FADD(dc_P[6][i],dc_Q[6][t]);
	}
}
__global__ void KFPoissonQnCalcW()
{
	int t=threadIdx.x+blockIdx.x*blockDim.x;
	int i;
	if(t<dc_Qnum[1])
	{
		i=dc_Qpos[1][t];
		dc_P[1][i]=FADD(dc_P[1][i],dc_Q[1][t]);
	}
	if(t<dc_Qnum[2])
	{
		i=dc_Qpos[2][t];
		dc_P[2][i]=FADD(dc_P[2][i],dc_Q[2][t]);
	}
	if(t<dc_Qnum[4])
	{
		i=dc_Qpos[4][t];
		dc_P[4][i]=FADD(dc_P[4][i],dc_Q[4][t]);
	}
	if(t<dc_Qnum[7])
	{
		i=dc_Qpos[7][t];
		dc_P[7][i]=FADD(dc_P[7][i],dc_Q[7][t]);
	}
}
int checkCUDAError(const char* msg);
int GetCUDADevStat();
extern "C" int DoPoissonSolverOnCudaFloat(PoissonSolverOnCudaStruct* PS, PoissonSolverOnCudaParamStruct CudaParm)
{
	printf("E1\n");
	GetCUDADevStat();
	DefClock0;
	int i,k;
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// Allocate vectors in device memory
	int ErrorCount=0;
	
	int GS_X=PS->GS[0];
	int GS_Y=PS->GS[1];
	int GS_Z=PS->GS[2];
	int GS_XY=GS_X*GS_Y;
	int GS_XYZ=GS_X*GS_Y*GS_Z;
	
	float om1 = 1.0-PS->Relaxation;
	float om2d6 = PS->Relaxation/6.0;
	hipMemcpyToSymbol(HIP_SYMBOL(dc_om1), &om1, sizeof(float), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL(dc_om2d6), &om2d6, sizeof(float), 0, hipMemcpyHostToDevice );
	
	printf("GS=[%d,%d,%d]=%d\n",GS_X,GS_Y,GS_Z,GS_XYZ);
	
	//Start Clock for GPU
	StartClock0;
	
	int BS_X=CudaParm.BS_X;
	int BS_Y=CudaParm.BS_Y;
	int BS_Z=CudaParm.BS_Z;
	int BS_XY=BS_X*BS_Y;
	int BS_XYZ=BS_X*BS_Y*BS_Z;
	
	dim3 dimBlock(BS_X,BS_Y,BS_Z);
	dim3 dimGridVirt(GS_X/BS_X/2, GS_Y/BS_Y/2, GS_Z/BS_Z/2);
	//d_P* is store in pitched array x has 16 for CUDA and Y/Z is +2 for BC
	dim3 spltGSWBC(PS->spltGSWBC[0],PS->spltGSWBC[1],PS->spltGSWBC[2]);
	dim3 dimGrid(dimGridVirt.x*dimGridVirt.y, dimGridVirt.z,1);
	
	
	int pitchX=spltGSWBC.x;
	int pitchXY=spltGSWBC.x*spltGSWBC.y;
	
	printf("dimBlock [%d,%d,%d]\n",dimBlock.x,dimBlock.y,dimBlock.z);
	printf("dimGrid [%d,%d,%d]\n",dimGrid.x,dimGrid.y,dimGrid.z);
	printf("dimGridVirt [%d,%d,%d]\n",dimGridVirt.x,dimGridVirt.y,dimGridVirt.z);
	printf("spltGSWBC [%d,%d,%d]\n",spltGSWBC.x,spltGSWBC.y,spltGSWBC.z);
	
	printf("Total number of threads %d\n",dimGrid.x*dimGrid.y*dimGrid.z*dimBlock.x*dimBlock.y*dimBlock.z);
	
	//fill dc_lookupVirGrid
	int  h_lookupVirGrid[dc_lookupVirGridSIZE];
	if(dc_lookupVirGridSIZE<dimGrid.x)
	{
		printf("ERROR: dc_lookupVirGridSIZE is smaller then dimGrid.x, make it at least %d\n",dimGrid.x);
		return 1;
	}
	int VirXblock,VirYblock;
	for(i=0;i<dimGrid.x;i++)
	{
		VirXblock = (i%dimGridVirt.x);
		VirYblock = (i/dimGridVirt.x);
		h_lookupVirGrid[i]=VirXblock*BS_X+(VirYblock*BS_Y + 1)*pitchX+pitchXY;
	}
	hipMemcpyToSymbol(HIP_SYMBOL(dc_lookupVirGrid), h_lookupVirGrid, dimGrid.x*sizeof(int), 0, hipMemcpyHostToDevice );
	
	
	int GS_XYZsplit = spltGSWBC.x*spltGSWBC.y*spltGSWBC.z;
	int sizeGS_XYZsplit = GS_XYZsplit*sizeof(float);
	
	//allocate and copy to device lin-array
	float* d_P[8];
	for(i=0;i<8;i++)
		hipMalloc((void**)&d_P[i], sizeGS_XYZsplit);
	for(i=0;i<8;i++)
		hipMemcpy(d_P[i], PS->P[i], sizeGS_XYZsplit, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(dc_P), d_P, 8*sizeof(float*), 0, hipMemcpyHostToDevice );
	//charges
	float* d_Q[8];
	int* d_Qpos[8];
	float* d_Qmult[8];
	int Qmax=0;
	for(i=0;i<8;i++)
	{
		d_Q[i]=NULL;
		d_Qpos[i]=NULL;
		d_Qmult[i]=NULL;
		if(PS->Qnum[i]>Qmax)Qmax=PS->Qnum[i];
		if(PS->Qnum[i]>0)
		{
			hipMalloc((void**)&d_Q[i], PS->Qnum[i]*sizeof(float));
			hipMalloc((void**)&d_Qpos[i], PS->Qnum[i]*sizeof(int));
			hipMalloc((void**)&d_Qmult[i], PS->Qnum[i]*sizeof(int));
			
			hipMemcpy(d_Q[i], PS->Q[i], PS->Qnum[i]*sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(d_Qpos[i], PS->Qpos[i], PS->Qnum[i]*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(d_Qmult[i], PS->Qmult[i], PS->Qnum[i]*sizeof(int), hipMemcpyHostToDevice);
		}
	}
	hipMemcpyToSymbol(HIP_SYMBOL(dc_Q), d_Q, 8*sizeof(float*), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL(dc_Qnum), PS->Qnum, 8*sizeof(int), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL(dc_Qpos), d_Qpos, 8*sizeof(int*), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL(dc_Qmult), d_Qmult, 8*sizeof(int*), 0, hipMemcpyHostToDevice );
	int Qblock=CudaParm.Qblock;
	int QGrid=Qmax/Qblock;
	if(Qmax%Qblock!=0)QGrid++;
	printf("Qmax=%d Qblock=%d QGrid=%d\n",Qmax,Qblock,QGrid);
	//Diel Border
	int d_DBNum[8];
	float* d_DielMult[48];
	int* d_DBPos[8];
	int DBmax=0;
	for(i=0;i<8;i++)
	{
		d_DBPos[i]=NULL;
		for(k=0;k<6;k++)
			d_DielMult[i*6+k]=NULL;
		if(PS->DielBordNum[i]>DBmax)DBmax=PS->DielBordNum[i];
		if(PS->DielBordNum[i]>0)
		{
			hipMalloc((void**)&d_DBPos[i], PS->DielBordNum[i]*sizeof(int));
			hipMemcpy(d_DBPos[i], PS->DielBordPos[i], PS->DielBordNum[i]*sizeof(int), hipMemcpyHostToDevice);
			for(k=0;k<6;k++)
			{
				hipMalloc((void**)&d_DielMult[i*6+k], PS->DielBordNum[i]*sizeof(float));
				hipMemcpy(d_DielMult[i*6+k], PS->DielMult[i][k], PS->DielBordNum[i]*sizeof(float), hipMemcpyHostToDevice);
			}
		}
	}
	hipMemcpyToSymbol(HIP_SYMBOL(dc_DBnum), PS->DielBordNum, 8*sizeof(int), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL(dc_DBpos), d_DBPos, 8*sizeof(int*), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL(dc_DielMult), d_DielMult, 48*sizeof(float*), 0, hipMemcpyHostToDevice );
	
	
	int DBblock=CudaParm.DBblock;
	int DBGrid=DBmax/DBblock;
	if(DBmax%DBblock!=0)DBGrid++;
	printf("DBmax=%d DBblock=%d DBGrid=%d\n",DBmax,DBblock,DBGrid);
	//do loop
	hipEventRecord( start, 0 );
	int j;
	double totalEnergy,dtmp1;
	double fpoh=4.0*M_PI*PS->GridScale;
	int AvrTECount=0;
	int CollectingTEforAvr=0;
	GetCUDADevStat();
	for(int iteration=1;iteration<=PS->MaxIterations;iteration++)
	{//pitchXY*BS_Z
		KFLaplaceB<<<dimGrid, dimBlock>>>(d_P[0],d_P[1],d_P[2],d_P[3], d_P[4],d_P[5],d_P[6],d_P[7], om1,om2d6, pitchX,pitchXY,pitchXY*BS_Z,BS_X,BS_XY);
		hipDeviceSynchronize();
		ErrorCount+=1-checkCUDAError("cuda kernel running: KFLaplaceB");
		
		if(QGrid>0)
		{
			KFPoissonQB<<<QGrid, Qblock>>>();
			hipDeviceSynchronize();
			ErrorCount+=1-checkCUDAError("cuda kernel running: KFPoissonQB");
		}
		if(DBGrid>0)
		{
			KFPoissonDBB<<<DBGrid, DBblock>>>(d_P[0],d_P[1],d_P[2],d_P[3], d_P[4],d_P[5],d_P[6],d_P[7], om2d6, pitchX,pitchXY);
			hipDeviceSynchronize();
			ErrorCount+=1-checkCUDAError("cuda kernel running: KFPoissonDBB");
		}
		
		KFLaplaceW<<<dimGrid, dimBlock>>>(d_P[0],d_P[1],d_P[2],d_P[3], d_P[4],d_P[5],d_P[6],d_P[7], om1,om2d6, pitchX,pitchXY,pitchXY*BS_Z,BS_X,BS_XY);
		hipDeviceSynchronize();
		ErrorCount+=1-checkCUDAError("cuda kernel running: KFLaplaceW");
		if(QGrid>0)
		{
			KFPoissonQW<<<QGrid, Qblock>>>();
			hipDeviceSynchronize();
			ErrorCount+=1-checkCUDAError("cuda kernel running: KFPoissonQW");
		}
		if(DBGrid>0)
		{
			KFPoissonDBW<<<DBGrid, DBblock>>>(d_P[0],d_P[1],d_P[2],d_P[3], d_P[4],d_P[5],d_P[6],d_P[7],om2d6, pitchX,pitchXY);
			hipDeviceSynchronize();
			ErrorCount+=1-checkCUDAError("cuda kernel running: KFPoissonDBW");
		}
		if(iteration%PS->ConvergenceCheck==0)
		{
			double OldTotalEnergy=totalEnergy;
			double totalChange;
			double relativeChange;
			double ConvFac;
			totalEnergy=0.0;
			for(i=0;i<8;i++)
			{
				hipMemcpy(PS->P[i], d_P[i], sizeGS_XYZsplit, hipMemcpyDeviceToHost);
				for(j=0;j<PS->Qnum[i];j++)
				{
					
					dtmp1=double(PS->P[i][PS->Qpos[i][j]])*double(PS->Q[i][j])/double(PS->Qmult[i][j]);
					
					totalEnergy+=dtmp1;
				}
			}
			totalEnergy=totalEnergy/(fpoh*2.0);
			
			totalChange=totalEnergy-OldTotalEnergy;
			relativeChange=totalChange/totalEnergy;
			
			printf("<PoissonIterations Nit=\"%8d\" E=\"%20.16e\" dE=\"%.4e\" rel.E=\"%.4e\" ConvFac=\"%.4e\"/>\n", iteration, totalEnergy, totalChange, relativeChange,ConvFac);
			if(PS->Tolerance!=0.0 && CollectingTEforAvr==0)
			{
				if(fabs(relativeChange)<=PS->Tolerance)
				{
					if(PS->AvrOverChecks>0)
					{
						printf("Solver has reached the requiered tolerance level\n");
						printf("Collecting total energies for final result\n");
						CollectingTEforAvr=1;
						PS->TotalEnergy=totalEnergy;
						PS->TEavr=0.0;
						PS->stdevTE=0.0;
						AvrTECount=0;
					}
					else
					{
						printf("Solver has reached the requiered tolerance level\n");
						break;
					}
				}
			}
			if(CollectingTEforAvr)
			{
				PS->TEavr+=totalEnergy;
				PS->stdevTE+=(totalEnergy-PS->TotalEnergy)*(totalEnergy-PS->TotalEnergy);
				AvrTECount++;
				if(AvrTECount==PS->AvrOverChecks)
				{
					printf("Finished collecting total energies for final result\n");
					PS->TEavr/=AvrTECount;
					PS->stdevTE/=AvrTECount;
					PS->stdevTE=PS->stdevTE-(PS->TEavr-PS->TotalEnergy)*(PS->TEavr-PS->TotalEnergy);
					//printf("TE=%.7f\tTEavr=%.7f\tstdevTE=%.3g\n",PS->TotalEnergy,PS->TEavr,PS->stdevTE);
					if(PS->stdevTE>0.0)
						PS->stdevTE=sqrt(PS->stdevTE);
					else
						PS->stdevTE=0.0;
					PS->TotalEnergy=totalEnergy;
					printf("TE=%.7f\tTEavr=%.7f\tstdevTE=%.3g\n",PS->TotalEnergy,PS->TEavr,PS->stdevTE);
					break;
				}
			}
		}
		if(ErrorCount)
			break;
	}
	GetCUDADevStat();
	ErrorCount+=1-checkCUDAError("cuda kernel running");
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time, start, stop );
	printf("Time on iterations: %e\n",time);
	time/=1000;//time in s
	printf("\tfor [%d,%d,%d] load store cycle: %f iter/s\n",GS_X,GS_Y,GS_Z,float(PS->MaxIterations)/time);
	//printf("block QGridQ time iter/s %d %d %g %g\n",Qblock,QGrid,time,float(PS->MaxIterations)/time);
	//copy from device lin-array		
	for(i=0;i<8;i++)
		hipMemcpy(PS->P[i], d_P[i], sizeGS_XYZsplit, hipMemcpyDeviceToHost);
	
	//free stuff
	for(i=0;i<8;i++)
	{
		if(PS->DielBordNum[i]>0)
		{
			for(k=0;k<6;k++)
			{
				hipFree(d_DielMult[i*6+k]);
			}
			hipFree(d_DBPos[i]);
		}
	}
	for(i=0;i<8;i++)
	{
		if(PS->Qnum[i]>0)
		{
			hipFree(d_Qmult[i]);
			hipFree(d_Qpos[i]);
			hipFree(d_Q[i]);
		}
	}
	for(i=0;i<8;i++)
		hipFree(d_P[i]);
	
	StopClockWMes0("GPU");
	GetCUDADevStat();
	if(ErrorCount)
	{
		printf("During GPU accelerated calculations found %d errors\n",ErrorCount);
		return 0;
	}
	return 1;
}
int checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) 
	{
		fprintf(stderr, "CUDA ERROR: %s: %s.\n", msg, 
						hipGetErrorString( err) );
		return 0;
	}
	return 1;
}
int GetCUDADevStat()
{
	printf("########################################\n");
	int device;
	hipGetDevice(&device);
			
	//hipSetDevice(device);
	hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, device);
	int countDev;
	hipGetDeviceCount(&countDev);
	printf("Totally %d CUDA devices present\n",countDev);
	printf("running on GPU #%d: %s\n", device,properties.name);
	printf("clockRate=%dkHz\n",properties.clockRate);
	printf("computeMode=%d\n",properties.computeMode);
	printf("Major compute capability: %d\n",properties.major);
	printf("Minor compute capability: %d\n",properties.minor);
	printf("Number of multiprocessors on device.: %d\n",properties.multiProcessorCount);
	printf("Maximum pitch in bytes allowed by memory copies: %d\n",properties.memPitch);
	printf("32-bit registers available per block: %d\n",properties.regsPerBlock);
	printf("Shared memory available per block in bytes: %d\n",properties.sharedMemPerBlock);
	printf("Constant memory available on device in bytes: %d\n",properties.totalConstMem);
	printf("Global memory available on device in bytes: %d\n",properties.totalGlobalMem);
	printf("Warp size in threads: %d\n",properties.warpSize);
	//hipDevice_t device;
	//hipDeviceGet(&device,  ordinal);
	//unsigned int totmem;
	//hipDeviceTotalMem(&totmem, device);
	//printf("memory available on device in bytes: %d\n",totmem);
	unsigned int free, total;
	hipMemGetInfo(&free, &total);
	if (properties.totalGlobalMem >= 1024*1024*1024) {
		printf("Total GPU Memory: %.4f GB\n", properties.totalGlobalMem/(1024.f*1024.f*1024.f) );
	} else {
		printf("Total GPU Memory: %.4f MB\n", properties.totalGlobalMem/(1024.f*1024.f) );
	}
	unsigned int free_mem,total_mem, used_mem;
	hipMemGetInfo( &free_mem, &total_mem );
	used_mem = total_mem-free_mem;
	printf("#CDS1 total mem: %0.3f MB, free: %0.3f MB, used : %0.3f MB\n",
		((double)total_mem)/1024.0/1024.0,
		((double)free_mem )/1024.0/1024.0,
		((double)used_mem )/1024.0/1024.0 );
	printf("#CDS2 total mem: %d, free: %d, used : %d\n",free_mem,total_mem, used_mem);
	printf("########################################\n");
	return 1;
}

#endif
