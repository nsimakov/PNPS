#include "hip/hip_runtime.h"
#ifndef _CUDADOUBLE_SOLVER_CU_
#define _CUDADOUBLE_SOLVER_CU_

#include <stdio.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>

#include "pnpstructs.h"
//#include "pnpdebug.h"

#define DefClock0 clock_t time0;timeval tvtime0,tvtime1;
#define StartClock0 time0=clock ();gettimeofday(&tvtime0,NULL);
#define StopClock0 gettimeofday(&tvtime1,NULL);DbgPrint0("Time : %g s(CPU Time) %g  s(Wall Time)\n",((double)(clock ()-time0))/CLOCKS_PER_SEC,double(tvtime1.tv_sec)+(double(tvtime1.tv_usec)/1000000.0)-double(tvtime0.tv_sec)-(double(tvtime0.tv_usec)/1000000.0));
#define StopClockWMes0(Massege) gettimeofday(&tvtime1,NULL);printf("Time for %s is %.5g s (CPU Time) %g  s(Wall Time)\n",(Massege),((double)(clock ()-time0))/CLOCKS_PER_SEC,double(tvtime1.tv_sec)+(double(tvtime1.tv_usec)/1000000.0)-double(tvtime0.tv_sec)-(double(tvtime0.tv_usec)/1000000.0));


#define FMUL __dmul_rn
#define FADD __dadd_rn
#define FMAF __dmaf_rn

__constant__ double  dc_om1;
__constant__ double  dc_om2d6;
__constant__ double* dc_P[8];
__constant__ int dc_Qnum[8];
__constant__ double* dc_Q[8];
__constant__ double* dc_Qmult[8];
__constant__ int* dc_Qpos[8];

__constant__ int dc_DBnum[8];
__constant__ double* dc_DielMult[48];
__constant__ int* dc_DBpos[8];
#define dc_lookupVirGridSIZE 1024
__constant__ int  dc_lookupVirGrid[dc_lookupVirGridSIZE];


__global__ void KDLaplaceB(double* d_P0,double* d_P1,double* d_P2,double* d_P3,
																double* d_P4,double* d_P5,double* d_P6,double* d_P7,
																const double om1,const double om2d6,
								const int pitchX,const int pitchXY,const int pitchXY_BS_Z,const int BS_X,const int BS_XY)
{
// 	int VirXblock = (blockIdx.x%VirGridX);
// 	int VirYblock = (blockIdx.x/VirGridX);
// 	int VirZblock = blockIdx.y;
// 	
// 	int tx = VirXblock*BS_X + threadIdx.x;
// 	int ty = VirYblock*BS_Y + threadIdx.y+1;
// 	int tz = VirZblock*BS_Z + threadIdx.z+1;
// 	
// 	int i=tx+ty*pitchX+tz*pitchXY;
	int t=threadIdx.x+threadIdx.y*pitchX+threadIdx.z*pitchXY;
	int i=dc_lookupVirGrid[blockIdx.x] + blockIdx.y*pitchXY_BS_Z + t;
	t=threadIdx.x+threadIdx.y*BS_X+threadIdx.z*BS_XY;
	
	double xP0,xP3,xP5,xP6;
	double yP0,yP3,yP5,yP6;
	double zP0,zP3,zP5,zP6;
	
	__shared__ double shP[256];
	
	
	//do over P1
	shP[t]=d_P1[i];
	__syncthreads();
	
	zP5=FADD(shP[t],d_P1[i+pitchXY]);
	yP3=FADD(shP[t],d_P1[i+pitchX]);
	if(threadIdx.x!=0)
		xP0=FADD(shP[t],shP[t-1]);
	else
		xP0=FADD(shP[t],d_P1[i-1]);
	__syncthreads();
	//do over P2
	shP[t]=d_P2[i];
	__syncthreads();
	
	zP6=FADD(shP[t],d_P2[i+pitchXY]);
	yP0=FADD(shP[t],d_P2[i-pitchX]);
	if(threadIdx.x!=blockDim.x-1)
		xP3=FADD(shP[t],shP[t+1]);
	else
		xP3=FADD(shP[t],d_P2[i+1]);
	__syncthreads();
	//do over P4
	shP[t]=d_P4[i];
	__syncthreads();
	zP0=FADD(shP[t],d_P4[i-pitchXY]);
	yP6=FADD(shP[t],d_P4[i+pitchX]);
	if(threadIdx.x!=blockDim.x-1)
		xP5=FADD(shP[t],shP[t+1]);
	else
		xP5=FADD(shP[t],d_P4[i+1]);
	__syncthreads();
	//do over P7
	shP[t]=d_P7[i];
	__syncthreads();
	zP3=FADD(shP[t],d_P7[i-pitchXY]);
	yP5=FADD(shP[t],d_P7[i-pitchX]);
	if(threadIdx.x!=0)
		xP6=FADD(shP[t],shP[t-1]);
	else
		xP6=FADD(shP[t],d_P7[i-1]);
	
	
	d_P0[i]=FADD(FMUL(om1,d_P0[i]),FMUL(om2d6,FADD(FADD(xP0,yP0),zP0)));
	d_P3[i]=FADD(FMUL(om1,d_P3[i]),FMUL(om2d6,FADD(FADD(xP3,yP3),zP3)));
	d_P5[i]=FADD(FMUL(om1,d_P5[i]),FMUL(om2d6,FADD(FADD(xP5,yP5),zP5)));
	d_P6[i]=FADD(FMUL(om1,d_P6[i]),FMUL(om2d6,FADD(FADD(xP6,yP6),zP6)));
}
__global__ void KDLaplaceW(double* d_P0,double* d_P1,double* d_P2,double* d_P3,
																double* d_P4,double* d_P5,double* d_P6,double* d_P7,
								const double om1,const double om2d6,
				const int pitchX,const int pitchXY,const int pitchXY_BS_Z,const int BS_X,const int BS_XY)
{
// 	int VirXblock = (blockIdx.x%VirGridX);
// 	int VirYblock = (blockIdx.x/VirGridX);
// 	int VirZblock = blockIdx.y;
// 	
// 	int tx = VirXblock*BS_X + threadIdx.x;
// 	int ty = VirYblock*BS_Y + threadIdx.y+1;
// 	int tz = VirZblock*BS_Z + threadIdx.z+1;
// 	
// 	int i=tx+ty*pitchX+tz*pitchXY;
	//int i=dc_lookupVirGrid[blockIdx.x] + blockIdx.y*pitchXY_BS_Z + threadIdx.x+threadIdx.y*pitchX+threadIdx.z*pitchXY;
	
	int t=threadIdx.x+threadIdx.y*pitchX+threadIdx.z*pitchXY;
	int i=dc_lookupVirGrid[blockIdx.x] + blockIdx.y*pitchXY_BS_Z + t;
	t=threadIdx.x+threadIdx.y*BS_X+threadIdx.z*BS_XY;
	
	double xP1,xP2,xP4,xP7;
	double yP1,yP2,yP4,yP7;
	double zP1,zP2,zP4,zP7;
	
	__shared__ double shP[256];
	
	//do dc_P[0]
	shP[t]=d_P0[i];
	__syncthreads();
	zP4=FADD(shP[t],d_P0[i+pitchXY]);
	yP2=FADD(shP[t],d_P0[i+pitchX]);
	if(threadIdx.x!=blockDim.x-1)
		xP1=FADD(shP[t],shP[t+1]);
	else
		xP1=FADD(shP[t],d_P0[i+1]);
	__syncthreads();
	//do d_P[3]
	shP[t]=d_P3[i];
	__syncthreads();
	zP7=FADD(shP[t],d_P3[i+pitchXY]);
	yP1=FADD(shP[t],d_P3[i-pitchX]);
	if(threadIdx.x!=0)
		xP2=FADD(shP[t],shP[t-1]);
	else
		xP2=FADD(shP[t],d_P3[i-1]);
	__syncthreads();
	//do d_P[5]
	shP[t]=d_P5[i];
	__syncthreads();
	zP1=FADD(shP[t],d_P5[i-pitchXY]);
	yP7=FADD(shP[t],d_P5[i+pitchX]);
	if(threadIdx.x!=0)
		xP4=FADD(shP[t],shP[t-1]);
	else
		xP4=FADD(shP[t],d_P5[i-1]);
	__syncthreads();
	//do d_P6
	shP[t]=d_P6[i];
	__syncthreads();
	zP2=FADD(shP[t],d_P6[i-pitchXY]);
	yP4=FADD(shP[t],d_P6[i-pitchX]);
	if(threadIdx.x!=blockDim.x-1)
		xP7=FADD(shP[t],shP[t+1]);
	else
		xP7=FADD(shP[t],d_P6[i+1]);
	
	//d_P6[i]=FADD(FMUL(om1,d_P6[i]),FMUL(om2d6,FADD(FADD(xP6,yP6),zP6)));
	
	d_P1[i]=FADD(FMUL(om1,d_P1[i]),FMUL(om2d6,FADD(FADD(xP1,yP1),zP1)));
	d_P2[i]=FADD(FMUL(om1,d_P2[i]),FMUL(om2d6,FADD(FADD(xP2,yP2),zP2)));
	d_P4[i]=FADD(FMUL(om1,d_P4[i]),FMUL(om2d6,FADD(FADD(xP4,yP4),zP4)));
	d_P7[i]=FADD(FMUL(om1,d_P7[i]),FMUL(om2d6,FADD(FADD(xP7,yP7),zP7)));
}
__global__ void KDPoissonQB()
{
	int t=threadIdx.x+blockIdx.x*blockDim.x;
	int i;
	if(t<dc_Qnum[0])
	{
		i=dc_Qpos[0][t];
		dc_P[0][i]=FADD(dc_P[0][i],dc_Q[0][t]);
	}
	if(t<dc_Qnum[3])
	{
		i=dc_Qpos[3][t];
		dc_P[3][i]=FADD(dc_P[3][i],dc_Q[3][t]);
	}
	if(t<dc_Qnum[5])
	{
		i=dc_Qpos[5][t];
		dc_P[5][i]=FADD(dc_P[5][i],dc_Q[5][t]);
	}
	if(t<dc_Qnum[6])
	{
		i=dc_Qpos[6][t];
		dc_P[6][i]=FADD(dc_P[6][i],dc_Q[6][t]);
	}
}
__global__ void KDPoissonQW()
{
	int t=threadIdx.x+blockIdx.x*blockDim.x;
	int i;
	if(t<dc_Qnum[1])
	{
		i=dc_Qpos[1][t];
		dc_P[1][i]=FADD(dc_P[1][i],dc_Q[1][t]);
	}
	if(t<dc_Qnum[2])
	{
		i=dc_Qpos[2][t];
		dc_P[2][i]=FADD(dc_P[2][i],dc_Q[2][t]);
	}
	if(t<dc_Qnum[4])
	{
		i=dc_Qpos[4][t];
		dc_P[4][i]=FADD(dc_P[4][i],dc_Q[4][t]);
	}
	if(t<dc_Qnum[7])
	{
		i=dc_Qpos[7][t];
		dc_P[7][i]=FADD(dc_P[7][i],dc_Q[7][t]);
	}
}
__global__ void KDPoissonDBB(double* d_P0,double* d_P1,double* d_P2,double* d_P3,
																double* d_P4,double* d_P5,double* d_P6,double* d_P7,
								const double om2d6,
				const int pitchX,const int pitchXY)
{
	int t=threadIdx.x+blockIdx.x*blockDim.x;
	int i;
	double xP,xM,yP,yM,zP,zM;
	if(t<dc_DBnum[0])
	{
		i=dc_DBpos[0][t];
		//P0 x
		xP=FMUL(dc_DielMult[PlusX][t],d_P1[i]);
		xM=FMUL(dc_DielMult[MinusX][t],d_P1[i-1]);
		//P0 y 
		yP=FMUL(dc_DielMult[PlusY][t],d_P2[i]);
		yM=FMUL(dc_DielMult[MinusY][t],d_P2[i-pitchX]);
		//P0 z
		zP=FMUL(dc_DielMult[PlusZ][t],d_P4[i]);
		zM=FMUL(dc_DielMult[MinusZ][t],d_P4[i-pitchXY]);
		
		xP=FADD(xM,xP);
		yP=FADD(yM,yP);
		zP=FADD(zM,zP);
		dc_P[0][i]=FADD(FADD(xP,yP),FADD(zP,dc_P[0][i]));
	}
	if(t<dc_DBnum[3])
	{
		i=dc_DBpos[3][t];
		//P3 x
		xM=FMUL(dc_DielMult[3*6+MinusX][t],d_P2[i]);
		xP=FMUL(dc_DielMult[3*6+PlusX][t],d_P2[i+1]);
		//P3 y
		yM=FMUL(dc_DielMult[3*6+MinusY][t],d_P1[i]);
		yP=FMUL(dc_DielMult[3*6+PlusY][t],d_P1[i+pitchX]);
		//P3 z
		zP=FMUL(dc_DielMult[3*6+PlusZ][t],d_P7[i]);
		zM=FMUL(dc_DielMult[3*6+MinusZ][t],d_P7[i-pitchXY]);
		
		xP=FADD(xM,xP);
		yP=FADD(yM,yP);
		zP=FADD(zM,zP);
		dc_P[3][i]=FADD(FADD(xP,yP),FADD(zP,dc_P[3][i]));
	}
	if(t<dc_DBnum[5])
	{
		i=dc_DBpos[5][t];
		//P5 x
		xM=FMUL(dc_DielMult[5*6+MinusX][t],d_P4[i]);
		xP=FMUL(dc_DielMult[5*6+PlusX][t],d_P4[i+1]);
		//P5 y
		yP=FMUL(dc_DielMult[5*6+PlusY][t],d_P7[i]);
		yM=FMUL(dc_DielMult[5*6+MinusY][t],d_P7[i-pitchX]);
		//P5 z
		zM=FMUL(dc_DielMult[5*6+MinusZ][t],d_P1[i]);
		zP=FMUL(dc_DielMult[5*6+PlusZ][t],d_P1[i+pitchXY]);
		
		xP=FADD(xM,xP);
		yP=FADD(yM,yP);
		zP=FADD(zM,zP);
		dc_P[5][i]=FADD(FADD(xP,yP),FADD(zP,dc_P[5][i]));
	}
	if(t<dc_DBnum[6])
	{
		i=dc_DBpos[6][t];
			//P6 x
		xP=FMUL(dc_DielMult[6*6+PlusX][t],d_P7[i]);
		xM=FMUL(dc_DielMult[6*6+MinusX][t],d_P7[i-1]);
		//P6 y
		yM=FMUL(dc_DielMult[6*6+MinusY][t],d_P4[i]);
		yP=FMUL(dc_DielMult[6*6+PlusY][t],d_P4[i+pitchX]);
		//P6 z
		zM=FMUL(dc_DielMult[6*6+MinusZ][t],d_P2[i]);
		zP=FMUL(dc_DielMult[6*6+PlusZ][t],d_P2[i+pitchXY]);
		
		xP=FADD(xM,xP);
		yP=FADD(yM,yP);
		zP=FADD(zM,zP);
		dc_P[6][i]=FADD(FADD(xP,yP),FADD(zP,dc_P[6][i]));
	}
}
__global__ void KDPoissonDBW(double* d_P0,double* d_P1,double* d_P2,double* d_P3,
																double* d_P4,double* d_P5,double* d_P6,double* d_P7,
								const double om2d6,
				const int pitchX,const int pitchXY)
{
	int t=threadIdx.x+blockIdx.x*blockDim.x;
	int i;
	double xP,xM,yP,yM,zP,zM;
	
	if(t<dc_DBnum[1])
	{
		i=dc_DBpos[1][t];
		//P1 x
		xM=FMUL(dc_DielMult[1*6+MinusX][t],d_P0[i]);
		xP=FMUL(dc_DielMult[1*6+PlusX][t],d_P0[i+1]);
		//P1 y
		yP=FMUL(dc_DielMult[1*6+PlusY][t],d_P3[i]);
		yM=FMUL(dc_DielMult[1*6+MinusY][t],d_P3[i-pitchX]);
		//P1 z
		zP=FMUL(dc_DielMult[1*6+PlusZ][t],d_P5[i]);
		zM=FMUL(dc_DielMult[1*6+MinusZ][t],d_P5[i-pitchXY]);
		
		xP=FADD(xM,xP);
		yP=FADD(yM,yP);
		zP=FADD(zM,zP);
		dc_P[1][i]=FADD(FADD(xP,yP),FADD(zP,dc_P[1][i]));
	}
	if(t<dc_DBnum[2])
	{
		i=dc_DBpos[2][t];
		//P2 x
		xP=FMUL(dc_DielMult[2*6+PlusX][t],d_P3[i]);
		xM=FMUL(dc_DielMult[2*6+MinusX][t],d_P3[i-1]);
		//P2 y
		yM=FMUL(dc_DielMult[2*6+MinusY][t],d_P0[i]);
		yP=FMUL(dc_DielMult[2*6+PlusY][t],d_P0[i+pitchX]);
		//P2 z
		zP=FMUL(dc_DielMult[2*6+PlusZ][t],d_P6[i]);
		zM=FMUL(dc_DielMult[2*6+MinusZ][t],d_P6[i-pitchXY]);
		
		xP=FADD(xM,xP);
		yP=FADD(yM,yP);
		zP=FADD(zM,zP);
		dc_P[2][i]=FADD(FADD(xP,yP),FADD(zP,dc_P[2][i]));
	}
	if(t<dc_DBnum[4])
	{
		i=dc_DBpos[4][t];
		//P4x
		xP=FMUL(dc_DielMult[4*6+PlusX][t],d_P5[i]);
		xM=FMUL(dc_DielMult[4*6+MinusX][t],d_P5[i-1]);
		//P4 y
		yP=FMUL(dc_DielMult[4*6+PlusY][t],d_P6[i]);
		yM=FMUL(dc_DielMult[4*6+MinusY][t],d_P6[i-pitchX]);
		//P4 z
		zM=FMUL(dc_DielMult[4*6+MinusZ][t],d_P0[i]);
		zP=FMUL(dc_DielMult[4*6+PlusZ][t],d_P0[i+pitchXY]);
		
		xP=FADD(xM,xP);
		yP=FADD(yM,yP);
		zP=FADD(zM,zP);
		dc_P[4][i]=FADD(FADD(xP,yP),FADD(zP,dc_P[4][i]));
	}
	if(t<dc_DBnum[7])
	{
		i=dc_DBpos[7][t];
		//P7 x
		xM=FMUL(dc_DielMult[7*6+MinusX][t],d_P6[i]);
		xP=FMUL(dc_DielMult[7*6+PlusX][t],d_P6[i+1]);
		//P7 y
		yM=FMUL(dc_DielMult[7*6+MinusY][t],d_P5[i]);
		yP=FMUL(dc_DielMult[7*6+PlusY][t],d_P5[i+pitchX]);
		//P7 z
		zM=FMUL(dc_DielMult[7*6+MinusZ][t],d_P3[i]);
		zP=FMUL(dc_DielMult[7*6+PlusZ][t],d_P3[i+pitchXY]);
		
		xP=FADD(xM,xP);
		yP=FADD(yM,yP);
		zP=FADD(zM,zP);
		dc_P[7][i]=FADD(FADD(xP,yP),FADD(zP,dc_P[7][i]));
	}
}

__global__ void KDPoissonQnCalcEB()
{
	int t=threadIdx.x+blockIdx.x*blockDim.x;
	int i;
	if(t<dc_Qnum[0])
	{
		i=dc_Qpos[0][t];
		dc_P[0][i]=FADD(dc_P[0][i],dc_Q[0][t]);
	}
	if(t<dc_Qnum[3])
	{
		i=dc_Qpos[3][t];
		dc_P[3][i]=FADD(dc_P[3][i],dc_Q[3][t]);
	}
	if(t<dc_Qnum[5])
	{
		i=dc_Qpos[5][t];
		dc_P[5][i]=FADD(dc_P[5][i],dc_Q[5][t]);
	}
	if(t<dc_Qnum[6])
	{
		i=dc_Qpos[6][t];
		dc_P[6][i]=FADD(dc_P[6][i],dc_Q[6][t]);
	}
}
__global__ void KDPoissonQnCalcW()
{
	int t=threadIdx.x+blockIdx.x*blockDim.x;
	int i;
	if(t<dc_Qnum[1])
	{
		i=dc_Qpos[1][t];
		dc_P[1][i]=FADD(dc_P[1][i],dc_Q[1][t]);
	}
	if(t<dc_Qnum[2])
	{
		i=dc_Qpos[2][t];
		dc_P[2][i]=FADD(dc_P[2][i],dc_Q[2][t]);
	}
	if(t<dc_Qnum[4])
	{
		i=dc_Qpos[4][t];
		dc_P[4][i]=FADD(dc_P[4][i],dc_Q[4][t]);
	}
	if(t<dc_Qnum[7])
	{
		i=dc_Qpos[7][t];
		dc_P[7][i]=FADD(dc_P[7][i],dc_Q[7][t]);
	}
}
int checkCUDAError(const char* msg);
int GetCUDADevStat();
extern "C" int DoPSolverOnCudaDouble(PoissonSolverOnCudaParamStruct* CudaParm,PSolverOnCudaStructDouble* PS)
{
	GetCUDADevStat();
	int i,k;
	printf("dimBlock [%d,%d,%d]\n",CudaParm->BS_X,CudaParm->BS_Y,CudaParm->BS_Z);
	DefClock0;
	
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	// Allocate vectors in device memory
	int ErrorCount=0;
	
	int GS_X=PS->GS[0];
	int GS_Y=PS->GS[1];
	int GS_Z=PS->GS[2];
	int GS_XY=GS_X*GS_Y;
	int GS_XYZ=GS_X*GS_Y*GS_Z;
	
	double om1 = 1.0-PS->Relaxation;
	double om2d6 = PS->Relaxation/6.0;
	hipMemcpyToSymbol(HIP_SYMBOL(dc_om1), &om1, sizeof(double), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL(dc_om2d6), &om2d6, sizeof(double), 0, hipMemcpyHostToDevice );
	
	printf("GS=[%d,%d,%d]=%d\n",GS_X,GS_Y,GS_Z,GS_XYZ);
	
	//Start Clock for GPU
	StartClock0;
	
	int BS_X=CudaParm->BS_X;
	int BS_Y=CudaParm->BS_Y;
	int BS_Z=CudaParm->BS_Z;
	int BS_XY=BS_X*BS_Y;
	int BS_XYZ=BS_X*BS_Y*BS_Z;
	
	dim3 dimBlock(BS_X,BS_Y,BS_Z);
	dim3 dimGridVirt(GS_X/BS_X/2, GS_Y/BS_Y/2, GS_Z/BS_Z/2);
	//d_P* is store in pitched array x has 16 for CUDA and Y/Z is +2 for BC
	dim3 spltGSWBC(PS->spltGSWBC[0],PS->spltGSWBC[1],PS->spltGSWBC[2]);
	dim3 dimGrid(dimGridVirt.x*dimGridVirt.y, dimGridVirt.z,1);
	
	
	int pitchX=spltGSWBC.x;
	int pitchXY=spltGSWBC.x*spltGSWBC.y;
	
	printf("dimBlock [%d,%d,%d]\n",dimBlock.x,dimBlock.y,dimBlock.z);
	printf("dimGrid [%d,%d,%d]\n",dimGrid.x,dimGrid.y,dimGrid.z);
	printf("dimGridVirt [%d,%d,%d]\n",dimGridVirt.x,dimGridVirt.y,dimGridVirt.z);
	printf("spltGSWBC [%d,%d,%d]\n",spltGSWBC.x,spltGSWBC.y,spltGSWBC.z);
	
	printf("Total number of threads %d\n",dimGrid.x*dimGrid.y*dimGrid.z*dimBlock.x*dimBlock.y*dimBlock.z);
	
	//fill dc_lookupVirGrid
	int  h_lookupVirGrid[dc_lookupVirGridSIZE];
	if(dc_lookupVirGridSIZE<dimGrid.x)
	{
		printf("ERROR: dc_lookupVirGridSIZE is smaller then dimGrid.x, make it at least %d\n",dimGrid.x);
		return 1;
	}
	int VirXblock,VirYblock;
	for(i=0;i<dimGrid.x;i++)
	{
		VirXblock = (i%dimGridVirt.x);
		VirYblock = (i/dimGridVirt.x);
		h_lookupVirGrid[i]=VirXblock*BS_X+(VirYblock*BS_Y + 1)*pitchX+pitchXY;
	}
	hipMemcpyToSymbol(HIP_SYMBOL(dc_lookupVirGrid), h_lookupVirGrid, dimGrid.x*sizeof(int), 0, hipMemcpyHostToDevice );
	
	
	int GS_XYZsplit = spltGSWBC.x*spltGSWBC.y*spltGSWBC.z;
	int sizeGS_XYZsplit = GS_XYZsplit*sizeof(double);
	
	//allocate and copy to device lin-array
	double* d_P[8];
	for(i=0;i<8;i++)
		hipMalloc((void**)&d_P[i], sizeGS_XYZsplit);
	for(i=0;i<8;i++)
		hipMemcpy(d_P[i], PS->P[i], sizeGS_XYZsplit, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(dc_P), d_P, 8*sizeof(double*), 0, hipMemcpyHostToDevice );
	//charges
	double* d_Q[8];
	int* d_Qpos[8];
	double* d_Qmult[8];
	int Qmax=0;
	for(i=0;i<8;i++)
	{
		d_Q[i]=NULL;
		d_Qpos[i]=NULL;
		d_Qmult[i]=NULL;
		if(PS->Qnum[i]>Qmax)Qmax=PS->Qnum[i];
		if(PS->Qnum[i]>0)
		{
			hipMalloc((void**)&d_Q[i], PS->Qnum[i]*sizeof(double));
			hipMalloc((void**)&d_Qpos[i], PS->Qnum[i]*sizeof(int));
			hipMalloc((void**)&d_Qmult[i], PS->Qnum[i]*sizeof(int));
			
			hipMemcpy(d_Q[i], PS->Q[i], PS->Qnum[i]*sizeof(double), hipMemcpyHostToDevice);
			hipMemcpy(d_Qpos[i], PS->Qpos[i], PS->Qnum[i]*sizeof(int), hipMemcpyHostToDevice);
			hipMemcpy(d_Qmult[i], PS->Qmult[i], PS->Qnum[i]*sizeof(int), hipMemcpyHostToDevice);
		}
	}
	hipMemcpyToSymbol(HIP_SYMBOL(dc_Q), d_Q, 8*sizeof(double*), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL(dc_Qnum), PS->Qnum, 8*sizeof(int), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL(dc_Qpos), d_Qpos, 8*sizeof(int*), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL(dc_Qmult), d_Qmult, 8*sizeof(int*), 0, hipMemcpyHostToDevice );
	int Qblock=CudaParm->Qblock;
	int QGrid=Qmax/Qblock;
	if(Qmax%Qblock!=0)QGrid++;
	printf("Qmax=%d Qblock=%d QGrid=%d\n",Qmax,Qblock,QGrid);
	//Diel Border
	int d_DBNum[8];
	double* d_DielMult[48];
	int* d_DBPos[8];
	int DBmax=0;
	for(i=0;i<8;i++)
	{
		d_DBPos[i]=NULL;
		for(k=0;k<6;k++)
			d_DielMult[i*6+k]=NULL;
		if(PS->DielBordNum[i]>DBmax)DBmax=PS->DielBordNum[i];
		if(PS->DielBordNum[i]>0)
		{
			hipMalloc((void**)&d_DBPos[i], PS->DielBordNum[i]*sizeof(int));
			hipMemcpy(d_DBPos[i], PS->DielBordPos[i], PS->DielBordNum[i]*sizeof(int), hipMemcpyHostToDevice);
			for(k=0;k<6;k++)
			{
				hipMalloc((void**)&d_DielMult[i*6+k], PS->DielBordNum[i]*sizeof(double));
				hipMemcpy(d_DielMult[i*6+k], PS->DielMult[i][k], PS->DielBordNum[i]*sizeof(double), hipMemcpyHostToDevice);
			}
		}
	}
	hipMemcpyToSymbol(HIP_SYMBOL(dc_DBnum), PS->DielBordNum, 8*sizeof(int), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL(dc_DBpos), d_DBPos, 8*sizeof(int*), 0, hipMemcpyHostToDevice );
	hipMemcpyToSymbol(HIP_SYMBOL(dc_DielMult), d_DielMult, 48*sizeof(double*), 0, hipMemcpyHostToDevice );
	
	
	int DBblock=CudaParm->DBblock;
	int DBGrid=DBmax/DBblock;
	if(DBmax%DBblock!=0)DBGrid++;
	printf("DBmax=%d DBblock=%d DBGrid=%d\n",DBmax,DBblock,DBGrid);
	//do loop
	hipEventRecord( start, 0 );
	int j;
	double totalEnergy,dtmp1;
	double fpoh=4.0*M_PI*PS->GridScale;
	GetCUDADevStat();
	for(int iteration=1;iteration<=PS->MaxIterations;iteration++)
	{//pitchXY*BS_Z
		KDLaplaceB<<<dimGrid, dimBlock>>>(d_P[0],d_P[1],d_P[2],d_P[3], d_P[4],d_P[5],d_P[6],d_P[7], om1,om2d6, pitchX,pitchXY,pitchXY*BS_Z,BS_X,BS_XY);
		hipDeviceSynchronize();
		ErrorCount+=1-checkCUDAError("cuda kernel running: KDLaplaceB");
		
		if(QGrid>0)
		{
			KDPoissonQB<<<QGrid, Qblock>>>();
			hipDeviceSynchronize();
			ErrorCount+=1-checkCUDAError("cuda kernel running: KDPoissonQB");
		}
		if(DBGrid>0)
		{
			KDPoissonDBB<<<DBGrid, DBblock>>>(d_P[0],d_P[1],d_P[2],d_P[3], d_P[4],d_P[5],d_P[6],d_P[7], om2d6, pitchX,pitchXY);
			hipDeviceSynchronize();
			ErrorCount+=1-checkCUDAError("cuda kernel running: KDPoissonDBB");
		}
		KDLaplaceW<<<dimGrid, dimBlock>>>(d_P[0],d_P[1],d_P[2],d_P[3], d_P[4],d_P[5],d_P[6],d_P[7], om1,om2d6, pitchX,pitchXY,pitchXY*BS_Z,BS_X,BS_XY);
		hipDeviceSynchronize();
		ErrorCount+=1-checkCUDAError("cuda kernel running: KDLaplaceW");
		if(QGrid>0)
		{
			KDPoissonQW<<<QGrid, Qblock>>>();
			hipDeviceSynchronize();
			ErrorCount+=1-checkCUDAError("cuda kernel running: KDPoissonQW");
		}
		if(DBGrid>0)
		{
			KDPoissonDBW<<<DBGrid, DBblock>>>(d_P[0],d_P[1],d_P[2],d_P[3], d_P[4],d_P[5],d_P[6],d_P[7],om2d6, pitchX,pitchXY);
			hipDeviceSynchronize();
			ErrorCount+=1-checkCUDAError("cuda kernel running: KDPoissonDBW");
		}
		
		if(iteration%PS->ConvergenceCheck==0)
		{
			double OldTotalEnergy=totalEnergy;
			double totalChange;
			double relativeChange;
			double ConvFac;
			totalEnergy=0.0;
			for(i=0;i<8;i++)
			{
				hipMemcpy(PS->P[i], d_P[i], sizeGS_XYZsplit, hipMemcpyDeviceToHost);
				for(j=0;j<PS->Qnum[i];j++)
				{
					
					dtmp1=double(PS->P[i][PS->Qpos[i][j]])*double(PS->Q[i][j])/double(PS->Qmult[i][j]);
					
					totalEnergy+=dtmp1;
				}
			}
			totalEnergy=totalEnergy/(fpoh*2.0);
			
			totalChange=totalEnergy-OldTotalEnergy;
			relativeChange=totalChange/totalEnergy;
			
			printf("<PoissonIterations Nit=\"%8d\" E=\"%20.16e\" dE=\"%.4e\" rel.E=\"%.4e\" ConvFac=\"%.4e\"/>\n", iteration, totalEnergy, totalChange, relativeChange,ConvFac);
			if(PS->Tolerance!=0.0)
			{
				if(fabs(relativeChange)<=PS->Tolerance)
				{
					printf("Solver has reached the requiered tolerance level\n");
					break;
				}
			}
		}
		if(ErrorCount)
			break;
	}
	GetCUDADevStat();
	ErrorCount+=1-checkCUDAError("cuda kernel running");
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );
	hipEventElapsedTime( &time, start, stop );
	printf("Time on iterations: %e\n",time);
	time/=1000;//time in s
	printf("\tfor [%d,%d,%d] load store cycle: %f iter/s\n",GS_X,GS_Y,GS_Z,double(PS->MaxIterations)/time);
	//printf("block QGridQ time iter/s %d %d %g %g\n",Qblock,QGrid,time,double(PS->MaxIterations)/time);
	//copy from device lin-array		
	for(i=0;i<8;i++)
		hipMemcpy(PS->P[i], d_P[i], sizeGS_XYZsplit, hipMemcpyDeviceToHost);
	
	//free stuff
	for(i=0;i<8;i++)
	{
		if(PS->DielBordNum[i]>0)
		{
			for(k=0;k<6;k++)
			{
				hipFree(d_DielMult[i*6+k]);
			}
			hipFree(d_DBPos[i]);
		}
	}
	for(i=0;i<8;i++)
	{
		if(PS->Qnum[i]>0)
		{
			hipFree(d_Qmult[i]);
			hipFree(d_Qpos[i]);
			hipFree(d_Q[i]);
		}
	}
	for(i=0;i<8;i++)
		hipFree(d_P[i]);
	
	StopClockWMes0("GPU");
	GetCUDADevStat();
	if(ErrorCount)
	{
		printf("During GPU accelerated calculations found %d errors\n",ErrorCount);
		return 0;
	}
	return 1;
}


#endif
